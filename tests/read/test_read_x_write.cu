/* Copyright 2023 Oscar Amoros Huguet

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <sstream>

#include "tests/testsCommon.cuh"
#include <cvGPUSpeedup.cuh>

#include <opencv2/cudaimgproc.hpp>

template <int I, int OC>
bool test_read_x_write(int NUM_ELEMS_X, int NUM_ELEMS_Y, cv::cuda::Stream& cv_stream, bool enabled) {
    std::stringstream error_s;
    bool passed = true;
    bool exception = false;

    if (enabled) {

        struct Parameters {
            cv::Scalar init;
            cv::Scalar val_sub;
            cv::Scalar val_mul;
            cv::Scalar val_div;
        };

        std::vector<Parameters> params = {
            {{2u}, {0.3f}, {1.f}, {3.2f}},
            {{2u, 37u}, {0.3f, 0.3f}, {1.f, 4.f}, {3.2f, 0.6f}},
            {{2u, 37u, 128u}, {0.3f, 0.3f, 0.3f}, {1.f, 4.f, 3.2f}, {3.2f, 0.6f, 11.8f}},
            {{2u, 37u, 128u, 20u}, {0.3f, 0.3f, 0.3f, 0.3f}, {1.f, 4.f, 3.2f, 0.5f}, {3.2f, 0.6f, 11.8f, 33.f}}
        };

        cv::Scalar val_init = params.at(CV_MAT_CN(OC)-1).init;
        cv::Scalar val_sub = params.at(CV_MAT_CN(OC)-1).val_sub;
        cv::Scalar val_mul = params.at(CV_MAT_CN(OC)-1).val_mul;
        cv::Scalar val_div = params.at(CV_MAT_CN(OC)-1).val_div;
        cv::Scalar val_add = params.at(CV_MAT_CN(OC)-1).val_div;

        try {
            cv::cuda::GpuMat d_input(NUM_ELEMS_Y, NUM_ELEMS_X, I, val_init);
            cv::cuda::GpuMat d_temp(NUM_ELEMS_Y, NUM_ELEMS_X, OC);
            cv::cuda::GpuMat d_output_cv(NUM_ELEMS_Y, NUM_ELEMS_X, OC);
            cv::cuda::GpuMat d_output_cvGS(NUM_ELEMS_Y, NUM_ELEMS_X, OC);

            cv::Mat h_cvResults(NUM_ELEMS_Y, NUM_ELEMS_X, OC);
            cv::Mat h_cvGSResults(NUM_ELEMS_Y, NUM_ELEMS_X, OC);

            // OpenCV version
            d_input.convertTo(d_temp, OC, cv_stream);
            cv::cuda::subtract(d_temp, val_sub, d_output_cv, cv::noArray(), -1, cv_stream);
            cv::cuda::multiply(d_output_cv, val_mul, d_temp, 1.0, -1, cv_stream);
            cv::cuda::divide(d_temp, val_div, d_output_cv, 1.0, -1, cv_stream);
            cv::cuda::add(d_output_cv, val_add, d_output_cv, cv::noArray(), -1, cv_stream);     

            // cvGPUSpeedup version
            cvGS::executeOperations(d_input, d_output_cvGS, cv_stream, 
                                            cvGS::convertTo<I, OC>(),
                                            cvGS::subtract<OC>(val_sub),
                                            cvGS::multiply<OC>(val_mul),
                                            cvGS::divide<OC>(val_div),
                                            cvGS::add<OC>(val_add));

            // Verify results
            d_output_cv.download(h_cvResults, cv_stream);
            d_output_cvGS.download(h_cvGSResults, cv_stream);

            cv_stream.waitForCompletion();

            passed = compareAndCheck<OC>(NUM_ELEMS_X, NUM_ELEMS_Y, h_cvResults, h_cvGSResults);
            
        } catch (const std::exception& e) {
            error_s << e.what();
            passed = false;
            exception = true;
        }

        if (!passed) {
            if (!exception) {
                std::stringstream ss;
                ss << "test_read_x_write<" << cvTypeToString<I>() << ", " << cvTypeToString<OC>();
                std::cout << ss.str() << "> failed!! RESULT ERROR: Some results do not match baseline." << std::endl;
            } else {
                std::stringstream ss;
                ss << "test_read_x_write<" << cvTypeToString<I>() << ", " << cvTypeToString<OC>();
                std::cout << ss.str() << "> failed!! EXCEPTION: " << error_s.str() << std::endl;
            }
        }

    }

    return passed;
}

template <int I, int O>
struct ChannelsCC {
    enum Values { input = I, output = O };
};

using CCChannels = fk::TypeList<ChannelsCC<3, 4>, ChannelsCC<4, 3>,
    ChannelsCC<3, 4>, ChannelsCC<4, 3>,
    ChannelsCC<3, 3>, ChannelsCC<4, 4>>;

template <cv::ColorConversionCodes CODE, int BASE>
bool testCvtColor(int NUM_ELEMS_X, int NUM_ELEMS_Y, cv::cuda::Stream& cv_stream, bool enabled) {
    using CCCType = fk::TypeAt_t<CODE, CCChannels>;
    constexpr int inputChannels = CCCType::Values::input;
    constexpr int outputChannels = CCCType::Values::output;
    constexpr int CV_INPUT_TYPE = CV_MAKETYPE(BASE, inputChannels);
    constexpr int CV_OUTPUT_TYPE = CV_MAKETYPE(BASE, outputChannels);

    std::stringstream error_s;
    bool passed = true;
    bool exception = false;

    if (enabled) {
        try {
            cv::Scalar initValue;

            if constexpr (inputChannels == 3) {
                initValue = cv::Scalar(1u, 2u, 3u);
            } else if constexpr (inputChannels == 4) {
                initValue = cv::Scalar(1u, 2u, 3u, 4u);
            }

            cv::cuda::GpuMat d_input(NUM_ELEMS_Y, NUM_ELEMS_X, CV_INPUT_TYPE, initValue);

            cv::cuda::GpuMat d_output(NUM_ELEMS_Y, NUM_ELEMS_X, CV_OUTPUT_TYPE);
            cv::cuda::GpuMat d_cvGSoutput(NUM_ELEMS_Y, NUM_ELEMS_X, CV_OUTPUT_TYPE);

            cv::cuda::cvtColor(d_input, d_output, CODE, 0, cv_stream);
            cvGS::executeOperations(d_input, d_cvGSoutput, cv_stream, cvGS::cvtColor<CODE, CV_INPUT_TYPE>());

            cv_stream.waitForCompletion();

            cv::Mat h_output(d_output);
            cv::Mat h_cvGSoutput(d_cvGSoutput);

            passed = compareAndCheck<CV_OUTPUT_TYPE>(NUM_ELEMS_X, NUM_ELEMS_Y, h_output, h_cvGSoutput);

        } catch (const std::exception& e) {
            error_s << e.what();
            passed = false;
            exception = true;
        }

        if (!passed) {
            if (!exception) {
                std::stringstream ss;
                ss << "testCvtColor";
                std::cout << ss.str() << " failed!! RESULT ERROR: Some results do not match baseline." << std::endl;
            }
            else {
                std::stringstream ss;
                ss << "testCvtColor";
                std::cout << ss.str() << " failed!! EXCEPTION: " << error_s.str() << std::endl;
            }
        }
    }

    return passed;
}

int main() {
    constexpr size_t NUM_ELEMS_X = 3840;
    constexpr size_t NUM_ELEMS_Y = 2160;

    cv::cuda::Stream cv_stream;

    cv::Mat::setDefaultAllocator(cv::cuda::HostMem::getAllocator(cv::cuda::HostMem::AllocType::PAGE_LOCKED));

    std::unordered_map<std::string, bool> results;
    results["test_read_x_write"] = true;
    results["testCvtColor"] = true;

    #define LAUNCH_TESTS(CV_INPUT, CV_OUTPUT) \
    results["test_read_x_write"] &= test_read_x_write<CV_INPUT, CV_OUTPUT>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, true);

    LAUNCH_TESTS(CV_8UC1, CV_32FC1)
    LAUNCH_TESTS(CV_8SC1, CV_32FC1)
    LAUNCH_TESTS(CV_16UC1, CV_32FC1)
    LAUNCH_TESTS(CV_16SC1, CV_32FC1)
    LAUNCH_TESTS(CV_32SC1, CV_32FC1)
    LAUNCH_TESTS(CV_32FC1, CV_32FC1)
    LAUNCH_TESTS(CV_8UC2, CV_32FC2)
    LAUNCH_TESTS(CV_8UC3, CV_32FC3)
    LAUNCH_TESTS(CV_8UC4, CV_32FC4)
    LAUNCH_TESTS(CV_8SC2, CV_32FC2)
    LAUNCH_TESTS(CV_8SC3, CV_32FC3)
    LAUNCH_TESTS(CV_8SC4, CV_32FC4)
    LAUNCH_TESTS(CV_16UC2, CV_32FC2)
    LAUNCH_TESTS(CV_16UC3, CV_32FC3)
    LAUNCH_TESTS(CV_16UC4, CV_32FC4)
    LAUNCH_TESTS(CV_16SC2, CV_32FC2)
    LAUNCH_TESTS(CV_16SC3, CV_32FC3)
    LAUNCH_TESTS(CV_16SC4, CV_32FC4)
    LAUNCH_TESTS(CV_32SC2, CV_32FC2)
    LAUNCH_TESTS(CV_32SC3, CV_32FC3)
    LAUNCH_TESTS(CV_32SC4, CV_32FC4)
    LAUNCH_TESTS(CV_32FC2, CV_64FC2)
    LAUNCH_TESTS(CV_32FC3, CV_64FC3)
    LAUNCH_TESTS(CV_32FC4, CV_64FC4)

#undef LAUNCH_TESTS

#define LAUNCH_TESTS(CV_COLOR_CONVERSION_CODE, CV_TYPE_DEPTH) \
results["testCvtColor"] &= testCvtColor<CV_COLOR_CONVERSION_CODE, CV_TYPE_DEPTH>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, true);

    LAUNCH_TESTS(cv::COLOR_BGR2BGRA, CV_8U)
    LAUNCH_TESTS(cv::COLOR_BGR2BGRA, CV_16U)
    LAUNCH_TESTS(cv::COLOR_BGR2BGRA, CV_32F)
    LAUNCH_TESTS(cv::COLOR_BGRA2BGR, CV_8U)
    LAUNCH_TESTS(cv::COLOR_BGRA2BGR, CV_16U)
    LAUNCH_TESTS(cv::COLOR_BGRA2BGR, CV_32F)
    LAUNCH_TESTS(cv::COLOR_BGR2RGBA, CV_8U)
    LAUNCH_TESTS(cv::COLOR_BGR2RGBA, CV_16U)
    LAUNCH_TESTS(cv::COLOR_BGR2RGBA, CV_32F)
    LAUNCH_TESTS(cv::COLOR_BGRA2RGB, CV_8U)
    LAUNCH_TESTS(cv::COLOR_BGRA2RGB, CV_16U)
    LAUNCH_TESTS(cv::COLOR_BGRA2RGB, CV_32F)
    LAUNCH_TESTS(cv::COLOR_BGR2RGB, CV_8U)
    LAUNCH_TESTS(cv::COLOR_BGR2RGB, CV_16U)
    LAUNCH_TESTS(cv::COLOR_BGR2RGB, CV_32F)
    LAUNCH_TESTS(cv::COLOR_BGRA2RGBA, CV_8U)
    LAUNCH_TESTS(cv::COLOR_BGRA2RGBA, CV_16U)
    LAUNCH_TESTS(cv::COLOR_BGRA2RGBA, CV_32F)

#undef LAUNCH_TESTS

    int returnValue = 0;
    for (const auto& [key, passed] : results) {
        if (passed) {
            std::cout << key << " passed!!" << std::endl;
        } else {
            std::cout << key << " failed!!" << std::endl;
            returnValue = -1;
        }
    }

    return returnValue;
}