#include "hip/hip_runtime.h"
﻿/* Copyright 2023 Mediaproduccion S.L.U. (Oscar Amoros Huguet)
   Copyright 2025 Oscar Amoros Huguet

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "tests/testsCommon.cuh"
#include <fused_kernel/algorithms/basic_ops/arithmetic.cuh>
#include <fused_kernel/fused_kernel.cuh>
#include <cvGPUSpeedup.cuh>


#include "tests/main.h"

bool testCircularBatchRead() {
    constexpr uint WIDTH = 32;
    constexpr uint HEIGHT = 32;
    constexpr uint BATCH = 15;
    constexpr uint FIRST = 4;

    hipStream_t stream;

    gpuErrchk(hipStreamCreate(&stream));

    std::vector<fk::Ptr2D<uchar3>> h_inputAllocations;

    std::vector<fk::Ptr2D<uchar3>> inputAllocations;
    std::array<fk::RawPtr<fk::_2D, uchar3>, BATCH> input;
    fk::Tensor<uchar3> output;
    fk::Tensor<uchar3> h_output;

    for (int i = 0; i < BATCH; i++) {
        fk::Ptr2D<uchar3> h_temp(WIDTH, HEIGHT, 0, fk::MemType::HostPinned);
        for (int y = 0; y < HEIGHT; y++) {
            for (int x = 0; x < WIDTH; x++) {
                const fk::Point p{ x, y, 0 };
                *fk::PtrAccessor<fk::_2D>::point(p, h_temp.ptr()) = fk::make_<uchar3>(i, i, i);
            }
        }
        h_inputAllocations.push_back(h_temp);
        fk::Ptr2D<uchar3> temp(WIDTH, HEIGHT);
        inputAllocations.push_back(temp);
        input[i] = temp;
        gpuErrchk(hipMemcpy2DAsync(temp.ptr().data, temp.dims().pitch, h_temp.ptr().data, h_temp.dims().pitch,
            h_temp.dims().width * sizeof(uchar3), h_temp.dims().height, hipMemcpyHostToDevice, stream));
    }
    output.allocTensor(WIDTH, HEIGHT, BATCH);
    h_output.allocTensor(WIDTH, HEIGHT, BATCH, 1, fk::MemType::HostPinned);

    fk::Read<fk::CircularBatchRead<fk::Ascendent, fk::PerThreadRead<fk::_2D, uchar3>, BATCH>> circularBatchRead;
    circularBatchRead.params.first = FIRST;
    for (int i = 0; i < BATCH; i++) {
        circularBatchRead.params.opData[i].params = input[i];
    }
    fk::WriteInstantiableOperation<fk::PerThreadWrite<fk::_3D, uchar3>> write3D{ {output} };

    fk::executeOperations(stream, circularBatchRead, write3D);

    gpuErrchk(hipMemcpyAsync(h_output.ptr().data, output.ptr().data, output.sizeInBytes(), hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipStreamSynchronize(stream));

    bool correct = true;
    for (int z = 0; z < BATCH; z++) {
        for (int y = 0; y < HEIGHT; y++) {
            for (int x = 0; x < WIDTH; x++) {
                fk::Point p{ x, y, z };
                uchar3 res = *fk::PtrAccessor<fk::_3D>::point(p, h_output.ptr());
                uchar newZ = (z + FIRST);
                uchar3 gt  = newZ >= BATCH ? fk::make_set<uchar3>(newZ - BATCH) : fk::make_set<uchar3>(newZ);
                correct &= res.x == gt.x;
                correct &= res.y == gt.y;
                correct &= res.z == gt.z;
            }
        }
    }

    return correct;
}

struct OneToOne {
    constexpr static __device__ __forceinline__ uint at(const uint& zIdx) {
        return zIdx + 1;
    }
};

bool testDivergentBatch() {
    constexpr uint WIDTH = 32;
    constexpr uint HEIGHT = 32;
    constexpr uint BATCH = 2;
    constexpr uint VAL_SUM = 3;

    hipStream_t stream;

    gpuErrchk(hipStreamCreate(&stream));

    std::vector<fk::Ptr2D<uint>> h_inputAllocations;
    std::vector<fk::Ptr2D<uint>> inputAllocations;
    std::array<fk::RawPtr<fk::_2D, uint>, BATCH> input;
    fk::Tensor<uint> output;
    fk::Tensor<uint> h_output;
    fk::Tensor<uint> h_groundTruth;

    for (uint i = 0; i < BATCH; i++) {
        fk::Ptr2D<uint> h_temp(WIDTH, HEIGHT, 0, fk::MemType::HostPinned);
        fk::setTo(i, h_temp);
        h_inputAllocations.push_back(h_temp);
        fk::Ptr2D<uint> temp(WIDTH, HEIGHT);
        inputAllocations.push_back(temp);
        input[i] = temp;
        gpuErrchk(hipMemcpy2DAsync(temp.ptr().data, temp.ptr().dims.pitch,
                                    h_temp.ptr().data, h_temp.ptr().dims.pitch,
                                    h_temp.dims().width * sizeof(uint), h_temp.dims().height,
                                    hipMemcpyHostToDevice, stream));
    }

    output.allocTensor(WIDTH, HEIGHT, BATCH);
    h_output.allocTensor(WIDTH, HEIGHT, BATCH, 1, fk::MemType::HostPinned);
    h_groundTruth.allocTensor(WIDTH, HEIGHT, BATCH, 1, fk::MemType::HostPinned);

    for (int z = 0; z < BATCH; z++) {
        if (z == 0) {
            for (int y = 0; y < HEIGHT; y++) {
                for (int x = 0; x < HEIGHT; x++) {
                    const fk::Point p{x,y,z};
                    *fk::PtrAccessor<fk::_3D>::point(p, h_groundTruth.ptr()) = VAL_SUM;
                }
            }
        } else {
            for (int y = 0; y < HEIGHT; y++) {
                for (int x = 0; x < HEIGHT; x++) {
                    const fk::Point p{x, y, z};
                    *fk::PtrAccessor<fk::_3D>::point(p, h_groundTruth.ptr()) = z;
                }
            }
        }
    }

    auto opSeq1 = fk::buildOperationSequence(fk::Read<fk::PerThreadRead<fk::_2D, uint>> { input[0] },
                                             fk::Binary<fk::Add<uint>> {VAL_SUM},
                                             fk::Write<fk::PerThreadWrite<fk::_3D, uint>> { output.ptr() });
    auto opSeq2 = fk::buildOperationSequence(fk::Read<fk::PerThreadRead<fk::_2D, uint>> { input[1] },
                                             fk::Write<fk::PerThreadWrite<fk::_3D, uint>> { output.ptr() });

    const dim3 block = dim3(std::min(static_cast<int>(inputAllocations[0].dims().width), 32),
                      std::min(static_cast<int>(inputAllocations[0].dims().height), 8));
    const dim3 grid{ (uint)ceil((float)WIDTH / (float)block.x), (uint)ceil((float)HEIGHT / (float)block.y), BATCH };
    fk::launchDivergentBatchTransformDPP_Kernel<fk::ParArch::GPU_NVIDIA, OneToOne><<<grid, block, 0, stream>>>(opSeq1, opSeq2);

    gpuErrchk(hipMemcpyAsync(h_output.ptr().data, output.ptr().data, output.sizeInBytes(), hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipStreamSynchronize(stream));

    bool correct = true;
    for (int z = 0; z < BATCH; z++) {
        for (int y = 0; y < HEIGHT; y++) {
            for (int x = 0; x < WIDTH; x++) {
                const fk::Point p{x, y, z};
                const uint gt = *fk::PtrAccessor<fk::_3D>::point(p, h_groundTruth.ptr());
                const uint res = *fk::PtrAccessor<fk::_3D>::point(p, h_output.ptr());
                correct &= gt == res;
            }
        }
    }

    return correct;
}

template <typename IT, typename OT>
bool testCircularTensor() {
    using TensorOT = typename fk::VectorTraits<OT>::base;
    constexpr uint BATCH = 15;
    constexpr uint WIDTH = 128;
    constexpr uint HEIGHT = 128;
    constexpr uint COLOR_PLANES = fk::cn<IT>;
    constexpr int ITERS = 100;

    fk::CircularTensor<TensorOT, COLOR_PLANES, BATCH, fk::CircularTensorOrder::NewestFirst, fk::ColorPlanes::Standard> myTensor(WIDTH, HEIGHT);
    fk::Tensor<TensorOT> h_myTensor(WIDTH, HEIGHT, BATCH, COLOR_PLANES, fk::MemType::HostPinned);
    fk::Ptr2D<IT> input(WIDTH, HEIGHT);
    fk::Ptr2D<IT> h_input(WIDTH, HEIGHT, 0, fk::MemType::HostPinned);

    fk::setTo(10.0f, h_myTensor);

    hipStream_t stream;
    gpuErrchk(hipStreamCreate(&stream));

    gpuErrchk(hipMemcpyAsync(myTensor.ptr().data, h_myTensor.ptr().data, myTensor.sizeInBytes(), hipMemcpyHostToDevice, stream));

    for (int i = 0; i < ITERS; i++) {
        fk::setTo(fk::make_<IT>(i + 1, i + 1, i + 1), input, stream);
        myTensor.update(stream, fk::Read<fk::PerThreadRead<fk::_2D, IT>> {input.ptr()},
                                fk::Unary<fk::SaturateCast<IT, OT>> {},
                                fk::Write<fk::TensorSplit<OT>> {myTensor.ptr()});
        gpuErrchk(hipStreamSynchronize(stream));
    }

    gpuErrchk(hipMemcpyAsync(h_myTensor.ptr().data, myTensor.ptr().data, myTensor.sizeInBytes(), hipMemcpyHostToDevice, stream));

    gpuErrchk(hipStreamSynchronize(stream));

    bool correct = true;
    for (int z = 0; z < BATCH; z++) {
        const TensorOT value = (TensorOT)(ITERS - z);
        for (int y = 0; y < HEIGHT; y++) {
            for (int x = 0; x < WIDTH; x++) {
                const fk::Point p{x, y, z};
                const TensorOT res = *fk::PtrAccessor<fk::_3D>::point(p, h_myTensor.ptr());
                correct &= value == res;
            }
        }
    }

    return correct;
}

template <int IT, int OT>
bool testCircularTensorcvGS() {
    using TensorOT = typename fk::VectorTraits<CUDA_T(OT)>::base;
    constexpr uint BATCH = 15;
    constexpr uint WIDTH =128;
    constexpr uint HEIGHT = 128;
    constexpr uint COLOR_PLANES = CV_MAT_CN(IT);
    constexpr int ITERS = 100;

    cvGS::CircularTensor<IT, CV_MAT_DEPTH(OT), COLOR_PLANES, BATCH, fk::CircularTensorOrder::NewestFirst> myTensor(WIDTH, HEIGHT);
    fk::Tensor<TensorOT> h_myTensor(WIDTH, HEIGHT, BATCH, COLOR_PLANES, fk::MemType::HostPinned);
    cv::cuda::GpuMat input(HEIGHT, WIDTH, IT);
    fk::Ptr2D<CUDA_T(IT)> h_input(WIDTH, HEIGHT, 0, fk::MemType::HostPinned);

    fk::setTo(10.f, h_myTensor);

    hipStream_t stream;
    gpuErrchk(hipStreamCreate(&stream));
    cv::cuda::Stream cv_stream = cv::cuda::StreamAccessor::wrapStream(stream);

    gpuErrchk(hipMemcpyAsync(myTensor.ptr().data, h_myTensor.ptr().data, myTensor.sizeInBytes(), hipMemcpyHostToDevice, stream));

    for (int i = 0; i < ITERS; i++) {
        fk::setTo(fk::make_<CUDA_T(IT)>(i + 1, i + 1, i + 1), h_input);
        gpuErrchk(hipMemcpy2DAsync(input.data, input.step,
                                    h_input.ptr().data, h_input.ptr().dims.pitch,
                                    h_input.ptr().dims.width * sizeof(CUDA_T(IT)),
                                    h_input.ptr().dims.height,
                                    hipMemcpyHostToDevice, stream));
        myTensor.update(cv_stream, input,
                        fk::Unary<fk::SaturateCast<CUDA_T(IT), CUDA_T(OT)>> {},
                        fk::Write<fk::TensorSplit<CUDA_T(OT)>> {myTensor.ptr()});
        gpuErrchk(hipStreamSynchronize(stream));
    }

    gpuErrchk(hipMemcpyAsync(h_myTensor.ptr().data, myTensor.ptr().data, myTensor.sizeInBytes(), hipMemcpyDeviceToHost, stream));

    gpuErrchk(hipStreamSynchronize(stream));

    bool correct = true;
    const size_t plane_pixels = h_myTensor.dims().width * h_myTensor.dims().height;
    for (int z = 0; z < BATCH; z++) {
        const TensorOT value = (TensorOT)(ITERS - z);
        for (int y = 0; y < HEIGHT; y++) {
            for (int x = 0; x < WIDTH; x++) {
                const fk::Point p{x, y, z};
                const TensorOT* workPlane = fk::PtrAccessor<fk::_3D>::point(p, h_myTensor.ptr());
                const TensorOT resX = *workPlane;
                correct &= value == resX;
                const TensorOT resY = *(workPlane + plane_pixels);
                correct &= value == resY;
                const TensorOT resZ = *(workPlane + (plane_pixels * 2));
                correct &= value == resZ;
            }
        }
    }

    return correct;
}

template <int IT, int OT>
bool testTransposedCircularTensorcvGS() {
    using TensorOT = typename fk::VectorTraits<CUDA_T(OT)>::base;
    constexpr uint BATCH = 15;
    constexpr uint WIDTH = 128;
    constexpr uint HEIGHT = 128;
    constexpr uint COLOR_PLANES = CV_MAT_CN(IT);
    constexpr int ITERS = 100;

    cvGS::CircularTensor<IT, CV_MAT_DEPTH(OT), COLOR_PLANES, BATCH, fk::CircularTensorOrder::NewestFirst, fk::ColorPlanes::Transposed> myTensor(WIDTH, HEIGHT);
    fk::TensorT<TensorOT> h_myTensor(WIDTH, HEIGHT, BATCH, COLOR_PLANES, fk::MemType::HostPinned);
    fk::TensorT<TensorOT> h_myInternalTensor(WIDTH, HEIGHT, BATCH, COLOR_PLANES, fk::MemType::HostPinned);
    cv::cuda::GpuMat input(HEIGHT, WIDTH, IT);
    fk::Ptr2D<CUDA_T(IT)> h_input(WIDTH, HEIGHT, 0, fk::MemType::HostPinned);

    fk::setTo(10.f, h_myTensor);

    hipStream_t stream;
    gpuErrchk(hipStreamCreate(&stream));
    cv::cuda::Stream cv_stream = cv::cuda::StreamAccessor::wrapStream(stream);

    gpuErrchk(hipMemcpyAsync(myTensor.ptr().data, h_myTensor.ptr().data, myTensor.sizeInBytes(), hipMemcpyHostToDevice, stream));

    for (int i = 0; i < ITERS; i++) {
        fk::setTo(fk::make_<CUDA_T(IT)>(i + 1, i + 1, i + 1), h_input);
        gpuErrchk(hipMemcpy2DAsync(input.data, input.step,
            h_input.ptr().data, h_input.ptr().dims.pitch,
            h_input.ptr().dims.width * sizeof(CUDA_T(IT)),
            h_input.ptr().dims.height,
            hipMemcpyHostToDevice, stream));
        myTensor.update(cv_stream, input,
            fk::Unary<fk::SaturateCast<CUDA_T(IT), CUDA_T(OT)>> {},
            fk::Write<fk::TensorTSplit<CUDA_T(OT)>> {myTensor.ptr()});
        gpuErrchk(hipStreamSynchronize(stream));
    }

    gpuErrchk(hipMemcpyAsync(h_myTensor.ptr().data, myTensor.ptr().data, myTensor.sizeInBytes(), hipMemcpyHostToDevice, stream));

    gpuErrchk(hipStreamSynchronize(stream));

    bool correct = true;
    const auto dims = h_myTensor.dims();
    const size_t plane_pixels = dims.width * dims.height;
    for (int cp = 0; cp < (int)dims.color_planes; cp++) {
        for (int y = 0; y < (int)dims.height; y++) {
            for (int z = 0; z < (int)BATCH; z++) {
                const auto* plane = fk::PtrAccessor<fk::T3D>::cr_point(fk::Point(0, 0, z), h_myTensor.ptr())
                    + (plane_pixels * dims.planes * cp);
                for (int x = 0; x < (int)dims.width; x++) {
                    correct &= ITERS - z == plane[x + (y * dims.width)];
                }
            }
        }
    }

    return correct;
}

template <int IT, int OT>
bool testTransposedOldestFirstCircularTensorcvGS() {
    using TensorOT = typename fk::VectorTraits<CUDA_T(OT)>::base;
    constexpr uint BATCH = 15;
    constexpr uint WIDTH = 128;
    constexpr uint HEIGHT = 128;
    constexpr uint COLOR_PLANES = CV_MAT_CN(IT);
    constexpr int ITERS = 100;

    cvGS::CircularTensor<IT, CV_MAT_DEPTH(OT), COLOR_PLANES, BATCH, fk::CircularTensorOrder::OldestFirst, fk::ColorPlanes::Transposed> myTensor(WIDTH, HEIGHT);
    fk::TensorT<TensorOT> h_myTensor(WIDTH, HEIGHT, BATCH, COLOR_PLANES, fk::MemType::HostPinned);
    fk::TensorT<TensorOT> h_myInternalTensor(WIDTH, HEIGHT, BATCH, COLOR_PLANES, fk::MemType::HostPinned);
    cv::cuda::GpuMat input(HEIGHT, WIDTH, IT);
    fk::Ptr2D<CUDA_T(IT)> h_input(WIDTH, HEIGHT, 0, fk::MemType::HostPinned);

    fk::setTo(10.f, h_myTensor);

    hipStream_t stream;
    gpuErrchk(hipStreamCreate(&stream));
    cv::cuda::Stream cv_stream = cv::cuda::StreamAccessor::wrapStream(stream);

    gpuErrchk(hipMemcpyAsync(myTensor.ptr().data, h_myTensor.ptr().data, myTensor.sizeInBytes(), hipMemcpyHostToDevice, stream));

    for (int i = 0; i < ITERS; i++) {
        fk::setTo(fk::make_<CUDA_T(IT)>(i + 1, i + 1, i + 1), h_input);
        gpuErrchk(hipMemcpy2DAsync(input.data, input.step,
            h_input.ptr().data, h_input.ptr().dims.pitch,
            h_input.ptr().dims.width * sizeof(CUDA_T(IT)),
            h_input.ptr().dims.height,
            hipMemcpyHostToDevice, stream));
        myTensor.update(cv_stream, input,
            fk::Unary<fk::SaturateCast<CUDA_T(IT), CUDA_T(OT)>> {},
            fk::Write<fk::TensorTSplit<CUDA_T(OT)>> {myTensor.ptr()});
        gpuErrchk(hipStreamSynchronize(stream));
    }

    gpuErrchk(hipMemcpyAsync(h_myTensor.ptr().data, myTensor.ptr().data, myTensor.sizeInBytes(), hipMemcpyHostToDevice, stream));

    gpuErrchk(hipStreamSynchronize(stream));

    bool correct = true;
    const auto dims = h_myTensor.dims();
    const size_t plane_pixels = dims.width * dims.height;
    for (int cp = 0; cp < (int)dims.color_planes; cp++) {
        for (int y = 0; y < (int)dims.height; y++) {
            for (int z = 0; z < (int)BATCH; z++) {
                const auto* plane = fk::PtrAccessor<fk::T3D>::cr_point(fk::Point(0, 0, z), h_myTensor.ptr())
                    + (plane_pixels * dims.planes * cp);
                for (int x = 0; x < (int)dims.width; x++) {
                    correct &= ITERS - (BATCH - z - 1) == plane[x + (y * dims.width)];
                }
            }
        }
    }

    return correct;
}

bool testOldestFirstCircularTensorcvGS_noSplit() {
    constexpr uint BATCH = 15;
    constexpr uint WIDTH = 128;
    constexpr uint HEIGHT = 128;
    // Number of planes representing one image
    constexpr uint COLOR_PLANES = 1; // This means that the image is in packed mode, each data element will contain all the color chanels for the same pixel
    constexpr int ITERS = 100;

    cvGS::CircularTensor<CV_8UC4, CV_32FC4, COLOR_PLANES, BATCH, fk::CircularTensorOrder::OldestFirst> myTensor(WIDTH, HEIGHT);
    using TensorType = CUDA_T(CV_32FC4);
    fk::Tensor<TensorType> h_myTensor(WIDTH, HEIGHT, BATCH, COLOR_PLANES, fk::MemType::HostPinned);
    fk::Tensor<TensorType> h_myInternalTensor(WIDTH, HEIGHT, BATCH, COLOR_PLANES, fk::MemType::HostPinned);
    cv::cuda::GpuMat input(HEIGHT, WIDTH, CV_8UC4);
    fk::Ptr2D<CUDA_T(CV_8UC4)> h_input(WIDTH, HEIGHT, 0, fk::MemType::HostPinned);

    fk::setTo(fk::make_set<float4>(10.0f), h_myTensor);

    hipStream_t stream;
    gpuErrchk(hipStreamCreate(&stream));
    cv::cuda::Stream cv_stream = cv::cuda::StreamAccessor::wrapStream(stream);

    gpuErrchk(hipMemcpyAsync(myTensor.ptr().data, h_myTensor.ptr().data, myTensor.sizeInBytes(), hipMemcpyHostToDevice, stream));

    for (int i = 0; i < ITERS; i++) {
        fk::setTo(fk::make_set<CUDA_T(CV_8UC4)>(i + 1), h_input);
        gpuErrchk(hipMemcpy2DAsync(input.data, input.step,
                                    h_input.ptr().data, h_input.ptr().dims.pitch,
                                    h_input.ptr().dims.width * sizeof(CUDA_T(CV_8UC4)),
                                    h_input.ptr().dims.height,
                                    hipMemcpyHostToDevice, stream));
        myTensor.update(cv_stream, input,
                        fk::Unary<fk::SaturateCast<CUDA_T(CV_8UC4), CUDA_T(CV_32FC4)>> {},
                        fk::Write<fk::TensorWrite<CUDA_T(CV_32FC4)>> {myTensor.ptr()});
                        gpuErrchk(hipStreamSynchronize(stream));
    }

    gpuErrchk(hipMemcpyAsync(h_myTensor.ptr().data, myTensor.ptr().data, myTensor.sizeInBytes(), hipMemcpyHostToDevice, stream));

    gpuErrchk(hipStreamSynchronize(stream));

    bool correct = true;
    const auto dims = h_myTensor.dims();
    const size_t plane_pixels = dims.width * dims.height;
    for (int cp = 0; cp < (int)dims.color_planes; cp++) {
        for (int y = 0; y < (int)dims.height; y++) {
            for (int z = 0; z < (int)BATCH; z++) {
                const float4* plane = fk::PtrAccessor<fk::_3D>::cr_point(fk::Point(0, 0, z), h_myTensor.ptr()) + (plane_pixels * dims.planes * cp);
                for (int x = 0; x < (int)dims.width; x++) {
                    const float4 groundTruth = fk::make_set<float4>(ITERS - (BATCH - z - 1));
                    const float4 computedValue = plane[x + (y * dims.width)];
                    correct &= abs(groundTruth.x - computedValue.x) < 0.00001f;
                    correct &= abs(groundTruth.y - computedValue.y) < 0.00001f;
                    correct &= abs(groundTruth.z - computedValue.z) < 0.00001f;
                    correct &= abs(groundTruth.w - computedValue.w) < 0.00001f;
                }
            }
        }
    }

    return correct;
}

int launch() {
    int returnValue = 0;
    if (testCircularBatchRead()) {
        std::cout << "testCircularBatchRead OK" << std::endl;
    } else {
        std::cout << "testCircularBatchRead Failed!" << std::endl;
        returnValue = -1;
    }
    if (testDivergentBatch()) {
        std::cout << "testDivergentBatch OK" << std::endl;
    } else {
        std::cout << "testDivergentBatch Failed!" << std::endl;
        returnValue = -1;
    }
    if (testCircularTensor<uchar3, float3>()) {
        std::cout << "testCircularTensor<uchar3, float3> OK" << std::endl;
    } else {
        std::cout << "testCircularTensor<uchar3, float3> Failed!" << std::endl;
        returnValue = -1;
    }
    if (testCircularTensorcvGS<CV_8UC3, CV_32FC3>()) {
        std::cout << "testCircularTensorcvGS<CV_8UC3, CV_32FC3> OK" << std::endl;
    } else {
        std::cout << "testCircularTensorcvGS<CV_8UC3, CV_32FC3> Failed!" << std::endl;
        returnValue = -1;
    }
    if (testTransposedCircularTensorcvGS<CV_8UC3, CV_32FC3>()) {
        std::cout << "testTransposedCircularTensorcvGS<CV_8UC3, CV_32FC3> OK" << std::endl;
    } else {
        std::cout << "testTransposedCircularTensorcvGS <CV_8UC3, CV_32FC3> Failed!" << std::endl;
        returnValue = -1;
    }
    if (testTransposedOldestFirstCircularTensorcvGS<CV_8UC3, CV_32FC3>()) {
        std::cout << "testTransposedOldestFirstCircularTensorcvGS<CV_8UC3, CV_32FC3> OK" << std::endl;
    } else {
        std::cout << "testTransposedOldestFirstCircularTensorcvGS <CV_8UC3, CV_32FC3> Failed!" << std::endl;
        returnValue = -1;
    }
    if (testOldestFirstCircularTensorcvGS_noSplit()) {
        std::cout << "testOldestFirstCircularTensorcvGS_noSplit OK" << std::endl;
    } else {
        std::cout << "testOldestFirstCircularTensorcvGS_noSplit Failed!" << std::endl;
        returnValue = -1;
    }

    return returnValue;
}