#include "hip/hip_runtime.h"
/* Copyright 2023 Oscar Amoros Huguet

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <iostream>

#include <fused_kernel/fused_kernel.cuh>

template <typename T>
bool testPtr_2D() {
    constexpr size_t width = 1920;
    constexpr size_t height = 1080;
    constexpr size_t width_crop = 300;
    constexpr size_t height_crop = 200;

    fk::Point startPoint = {100, 200};

    fk::Ptr2D<T> input(width, height);
    fk::Ptr2D<T> cropedInput = input.crop(startPoint, fk::PtrDims<fk::_2D>(width_crop, height_crop));
    fk::Ptr2D<T> output(width_crop, height_crop);
    fk::Ptr2D<T> outputBig(width, height);

    hipStream_t stream;
    gpuErrchk(hipStreamCreate(&stream));

    dim3 block2D(32,8);
    dim3 grid2D(std::ceil(width_crop / (float)block2D.x),
                std::ceil(height_crop / (float)block2D.y));
    dim3 grid2DBig(std::ceil(width / (float)block2D.x),
                   std::ceil(height / (float)block2D.y));

    fk::memory_write_scalar<fk::_2D, fk::perthread_write<fk::_2D, T>, T> opFinal_2D = { output };
    fk::memory_write_scalar<fk::_2D, fk::perthread_write<fk::_2D, T>, T> opFinal_2DBig = { outputBig };

    for (int i=0; i<100; i++) {
        fk::cuda_transform_<<<grid2D, block2D, 0, stream>>>(cropedInput.ptr(), opFinal_2D);
        fk::cuda_transform_<<<grid2DBig, block2D, 0, stream>>>(input.ptr(), opFinal_2DBig);
    }

    hipError_t err = hipStreamSynchronize(stream);

    // TODO: use some values and check results correctness

    if (err != hipSuccess) {
        return false;
    } else {
        return true;
    }
}

int main() {
    bool test2Dpassed = true;

    test2Dpassed &= testPtr_2D<uchar>();
    test2Dpassed &= testPtr_2D<uchar3>();
    test2Dpassed &= testPtr_2D<float>();
    test2Dpassed &= testPtr_2D<float3>();

    hipStream_t stream;
    gpuErrchk(hipStreamCreate(&stream));

    fk::Ptr2D<uchar> input(64,64);
    fk::Ptr2D<uint> output(64,64);
    
    fk::unary_operation_scalar<fk::unary_cast<uchar, uint>, uint> op = {};
    fk::memory_write_scalar<fk::_2D, fk::perthread_write<fk::_2D, uint>, uint> opFinal_2D = { output };

    fk::cuda_transform_<<<dim3(1,8),dim3(64,8),0,stream>>>(input.ptr(), op);

    gpuErrchk(hipStreamSynchronize(stream));

    if (test2Dpassed) {
        std::cout << "testPtr_2D Success!!" << std::endl; 
    } else {
        std::cout << "testPtr_2D Failed!!" << std::endl;
    }

    return 0;
}