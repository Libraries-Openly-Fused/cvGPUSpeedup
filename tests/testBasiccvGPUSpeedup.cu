#include "hip/hip_runtime.h"
/* Copyright 2023 Oscar Amoros Huguet

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <sstream>

#include <testUtils.h>
#include <cvGPUSpeedup.h>

#include <opencv2/core.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/imgproc.hpp>

template <int T>
bool checkResults(int NUM_ELEMS_X, int NUM_ELEMS_Y, cv::Mat& h_comparison1C) {
    cv::Mat h_comparison;
    cv::Mat maxError(NUM_ELEMS_Y, NUM_ELEMS_X, T, static_cast<BASE_CUDA_T(T)>(0.0001f));
    cv::compare(h_comparison1C, maxError, h_comparison, cv::CMP_LT);
    
    int errors = cv::countNonZero(h_comparison1C);
    return errors == 0;
}

template <int I, int OC>
void testSplitOutputOperation(int NUM_ELEMS_X, int NUM_ELEMS_Y, cv::cuda::Stream& cv_stream) {
    std::stringstream error;
    bool passed = true;

    struct Parameters {
        cv::Scalar init;
        cv::Scalar alpha;
        cv::Scalar val_sub;
        cv::Scalar val_div;
    };

    double alpha = 0.3;

    std::vector<Parameters> params = {
        {{2u}, {alpha}, {1.f}, {3.2f}},
        {{2u, 37u}, {alpha, alpha}, {1.f, 4.f}, {3.2f, 0.6f}},
        {{2u, 37u, 128u}, {alpha, alpha, alpha}, {1.f, 4.f, 3.2f}, {3.2f, 0.6f, 11.8f}},
        {{2u, 37u, 128u, 20u}, {alpha, alpha, alpha, alpha}, {1.f, 4.f, 3.2f, 0.5f}, {3.2f, 0.6f, 11.8f, 33.f}}
    };

    cv::Scalar val_init = params.at(CV_MAT_CN(OC)-1).init;
    cv::Scalar val_alpha = params.at(CV_MAT_CN(OC)-1).alpha;
    cv::Scalar val_sub = params.at(CV_MAT_CN(OC)-1).val_sub;
    cv::Scalar val_div = params.at(CV_MAT_CN(OC)-1).val_div;

    try {
        cv::cuda::GpuMat d_input(NUM_ELEMS_Y, NUM_ELEMS_X, I, val_init);
        cv::cuda::GpuMat d_temp(NUM_ELEMS_Y, NUM_ELEMS_X, OC);
        cv::cuda::GpuMat d_temp2(NUM_ELEMS_Y, NUM_ELEMS_X, OC);

        cv::Mat diff(NUM_ELEMS_Y, NUM_ELEMS_X, CV_MAT_DEPTH(OC));
        std::vector<cv::Mat> h_cvResults(CV_MAT_CN(OC));
        std::vector<cv::Mat> h_cvGSResults(CV_MAT_CN(OC));
        std::vector<cv::cuda::GpuMat> d_output_cv(CV_MAT_CN(OC));
        std::vector<cv::cuda::GpuMat> d_output_cvGS(CV_MAT_CN(OC));

        for (int i=0; i<CV_MAT_CN(I); i++) {
            d_output_cv.at(i).create(NUM_ELEMS_Y, NUM_ELEMS_X, CV_MAT_DEPTH(OC));
            h_cvResults.at(i).create(NUM_ELEMS_Y, NUM_ELEMS_X, CV_MAT_DEPTH(OC));
            d_output_cvGS.at(i).create(NUM_ELEMS_Y, NUM_ELEMS_X, CV_MAT_DEPTH(OC));
            h_cvGSResults.at(i).create(NUM_ELEMS_Y, NUM_ELEMS_X, CV_MAT_DEPTH(OC));
        }

        // OpenCV version
        d_input.convertTo(d_temp, OC, alpha, cv_stream);
        cv::cuda::subtract(d_temp, val_sub, d_temp2, cv::noArray(), -1, cv_stream);
        cv::cuda::divide(d_temp2, val_div, d_temp, 1.0, -1, cv_stream);
        cv::cuda::split(d_temp, d_output_cv, cv_stream);

        // cvGPUSpeedup version
        cvGS::executeOperations<I>(d_input, cv_stream, 
                                                cvGS::convertTo<I, OC>(),
                                                cvGS::multiply<OC>(val_alpha),
                                                cvGS::subtract<OC>(val_sub),
                                                cvGS::divide<OC>(val_div),
                                                cvGS::split<OC>(d_output_cvGS));

        // Looking at Nsight Systems, with an RTX A2000 12GB
        // Speedups are up to 7x, depending on the data type

        // Verify results
        for (int i=0; i<CV_MAT_CN(OC); i++) {
            d_output_cv.at(i).download(h_cvResults.at(i), cv_stream);
            d_output_cvGS.at(i).download(h_cvGSResults.at(i), cv_stream);
        }

        cv_stream.waitForCompletion();

        for (int i=0; i<CV_MAT_CN(OC); i++) {
            diff = cv::abs(h_cvResults.at(i) - h_cvGSResults.at(i));
            passed &= checkResults<CV_MAT_DEPTH(OC)>(NUM_ELEMS_X, NUM_ELEMS_Y, diff);
        }
    } catch (const std::exception& e) {
        error << e.what();
        passed = false;
    }

    std::stringstream ss;
    ss << "testSplitOutputOperation<" << cvTypeToString<I>() << ", " << cvTypeToString<OC>();

    if (passed) {
        std::cout << ss.str() << "> passed!!" << std::endl;
    } else {
        std::cout << ss.str() << "> failed!! ERROR: " << error.str() << std::endl;
    }
}

template <int I, int OC>
void testNoDefinedOutputOperation(int NUM_ELEMS_X, int NUM_ELEMS_Y, cv::cuda::Stream& cv_stream) {
    std::stringstream error;
    bool passed = true;

    struct Parameters {
        cv::Scalar init;
        cv::Scalar val_sub;
        cv::Scalar val_mul;
        cv::Scalar val_div;
    };

    std::vector<Parameters> params = {
        {{2u}, {0.3f}, {1.f}, {3.2f}},
        {{2u, 37u}, {0.3f, 0.3f}, {1.f, 4.f}, {3.2f, 0.6f}},
        {{2u, 37u, 128u}, {0.3f, 0.3f, 0.3f}, {1.f, 4.f, 3.2f}, {3.2f, 0.6f, 11.8f}},
        {{2u, 37u, 128u, 20u}, {0.3f, 0.3f, 0.3f, 0.3f}, {1.f, 4.f, 3.2f, 0.5f}, {3.2f, 0.6f, 11.8f, 33.f}}
    };

    cv::Scalar val_init = params.at(CV_MAT_CN(OC)-1).init;
    cv::Scalar val_sub = params.at(CV_MAT_CN(OC)-1).val_sub;
    cv::Scalar val_mul = params.at(CV_MAT_CN(OC)-1).val_mul;
    cv::Scalar val_div = params.at(CV_MAT_CN(OC)-1).val_div;
    cv::Scalar val_add = params.at(CV_MAT_CN(OC)-1).val_div;

    try {
        cv::cuda::GpuMat d_input(NUM_ELEMS_Y, NUM_ELEMS_X, I, val_init);
        cv::cuda::GpuMat d_temp(NUM_ELEMS_Y, NUM_ELEMS_X, OC);
        cv::cuda::GpuMat d_output_cv(NUM_ELEMS_Y, NUM_ELEMS_X, OC);
        cv::cuda::GpuMat d_output_cvGS(NUM_ELEMS_Y, NUM_ELEMS_X, OC);

        cv::Mat h_cvResults(NUM_ELEMS_Y, NUM_ELEMS_X, OC);
        cv::Mat h_cvGSResults(NUM_ELEMS_Y, NUM_ELEMS_X, OC);

        // OpenCV version
        d_input.convertTo(d_temp, OC, cv_stream);
        cv::cuda::subtract(d_temp, val_sub, d_output_cv, cv::noArray(), -1, cv_stream);
        cv::cuda::multiply(d_output_cv, val_mul, d_temp, 1.0, -1, cv_stream);
        cv::cuda::divide(d_temp, val_div, d_output_cv, 1.0, -1, cv_stream);
        cv::cuda::add(d_output_cv, val_add, d_output_cv, cv::noArray(), -1, cv_stream);     

        // cvGPUSpeedup version
        cvGS::executeOperations<I, OC>(d_input, d_output_cvGS, cv_stream, 
                                       cvGS::convertTo<I, OC>(),
                                       cvGS::subtract<OC>(val_sub),
                                       cvGS::multiply<OC>(val_mul),
                                       cvGS::divide<OC>(val_div),
                                       cvGS::add<OC>(val_add));

        // Looking at Nsight Systems, with an RTX A2000 12GB
        // Speedups are up to 7x, depending on the data type

        // Verify results
        d_output_cv.download(h_cvResults, cv_stream);
        d_output_cvGS.download(h_cvGSResults, cv_stream);

        cv_stream.waitForCompletion();

        cv::Mat diff = cv::abs(h_cvResults - h_cvGSResults);
        std::vector<cv::Mat> h_comparison1C(CV_MAT_CN(OC));
        cv::split(diff, h_comparison1C);

        for (int i=0; i<CV_MAT_CN(OC); i++) {
            passed &= checkResults<CV_MAT_DEPTH(OC)>(NUM_ELEMS_X, NUM_ELEMS_Y, h_comparison1C.at(i));
        }
    } catch (const std::exception& e) {
        error << e.what();
        passed = false;
    }

    std::stringstream ss;
    ss << "testNoDefinedOutputOperation<" << cvTypeToString<I>() << ", " << cvTypeToString<OC>();

    if (passed) {
        std::cout << ss.str() << "> passed!!" << std::endl;
    } else {
        std::cout << ss.str() << "> failed!! ERROR: " << error.str() << std::endl;
    }
}

#define LAUNCH_TESTS(CV_INPUT, CV_OUTPUT) \
testNoDefinedOutputOperation<CV_INPUT, CV_OUTPUT>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream); \
testSplitOutputOperation<CV_INPUT, CV_OUTPUT>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream);


int main() {
    constexpr size_t NUM_ELEMS_X = 3840;
    constexpr size_t NUM_ELEMS_Y = 2160;

    cv::cuda::Stream cv_stream;

    cv::Mat::setDefaultAllocator(cv::cuda::HostMem::getAllocator (cv::cuda::HostMem::AllocType::PAGE_LOCKED));

    testNoDefinedOutputOperation<CV_8UC1, CV_32FC1>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream);
    testNoDefinedOutputOperation<CV_8SC1, CV_32FC1>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream);
    testNoDefinedOutputOperation<CV_16UC1, CV_32FC1>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream);
    testNoDefinedOutputOperation<CV_16SC1, CV_32FC1>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream);
    testNoDefinedOutputOperation<CV_32SC1, CV_32FC1>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream);

    LAUNCH_TESTS(CV_8UC2, CV_32FC2)
    LAUNCH_TESTS(CV_8UC3, CV_32FC3)
    LAUNCH_TESTS(CV_8UC4, CV_32FC4)
    LAUNCH_TESTS(CV_8SC2, CV_32FC2)
    LAUNCH_TESTS(CV_8SC3, CV_32FC3)
    LAUNCH_TESTS(CV_8SC4, CV_32FC4)
    LAUNCH_TESTS(CV_16UC2, CV_32FC2)
    LAUNCH_TESTS(CV_16UC3, CV_32FC3)
    LAUNCH_TESTS(CV_16UC4, CV_32FC4)
    LAUNCH_TESTS(CV_16SC2, CV_32FC2)
    LAUNCH_TESTS(CV_16SC3, CV_32FC3)
    LAUNCH_TESTS(CV_16SC4, CV_32FC4)
    LAUNCH_TESTS(CV_32SC2, CV_32FC2)
    LAUNCH_TESTS(CV_32SC3, CV_32FC3)
    LAUNCH_TESTS(CV_32SC4, CV_32FC4)
    LAUNCH_TESTS(CV_32FC2, CV_64FC2)
    LAUNCH_TESTS(CV_32FC3, CV_64FC3)
    LAUNCH_TESTS(CV_32FC4, CV_64FC4)

    return 0;
}