#include "hip/hip_runtime.h"
/* Copyright 2023 Oscar Amoros Huguet

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <sstream>

#include <testUtils.h>
#include <cvGPUSpeedup.h>

#include <opencv2/core.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudawarping.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/imgproc.hpp>

template <int T>
bool checkResults(int NUM_ELEMS_X, int NUM_ELEMS_Y, cv::Mat& h_comparison1C) {
    cv::Mat h_comparison(NUM_ELEMS_Y, NUM_ELEMS_X, T);
    cv::Mat maxError(NUM_ELEMS_Y, NUM_ELEMS_X, T, 0.00001);
    cv::compare(h_comparison1C, maxError, h_comparison, cv::CMP_GT);

#ifdef CVGS_DEBUG
    for (int y=0; y<h_comparison1C.rows; y++) {
        for (int x=0; x<h_comparison1C.cols; x++) {
            CUDA_T(T) value = h_comparison1C.at<CUDA_T(T)>(y,x);
            if (value > 0.001) {
                std::cout << value << ", ";
            }
        }
        std::cout << std::endl;
    }
#endif
    
    int errors = cv::countNonZero(h_comparison);
    return errors == 0;
}

template <int T>
bool compareAndCheck(int NUM_ELEMS_X, int NUM_ELEMS_Y, cv::Mat& cvVersion, cv::Mat& cvGSVersion) {
    bool passed = true;
    cv::Mat diff = cv::abs(cvVersion - cvGSVersion);
    std::vector<cv::Mat> h_comparison1C(CV_MAT_CN(T));
    cv::split(diff, h_comparison1C);

    for (int i=0; i<CV_MAT_CN(T); i++) {
        passed &= checkResults<CV_MAT_DEPTH(T)>(NUM_ELEMS_X, NUM_ELEMS_Y, h_comparison1C.at(i));
    }
    return passed;
}

template <int I, int OC>
bool testSplitOutputOperation(int NUM_ELEMS_X, int NUM_ELEMS_Y, cv::cuda::Stream& cv_stream, bool enabled) {
    std::stringstream error_s;
    bool passed = true;
    bool exception = false;

    if (enabled) {
        struct Parameters {
            cv::Scalar init;
            cv::Scalar alpha;
            cv::Scalar val_sub;
            cv::Scalar val_div;
        };

        double alpha = 0.3;

        std::vector<Parameters> params = {
            {{2u}, {alpha}, {1.f}, {3.2f}},
            {{2u, 37u}, {alpha, alpha}, {1.f, 4.f}, {3.2f, 0.6f}},
            {{2u, 37u, 128u}, {alpha, alpha, alpha}, {1.f, 4.f, 3.2f}, {3.2f, 0.6f, 11.8f}},
            {{2u, 37u, 128u, 20u}, {alpha, alpha, alpha, alpha}, {1.f, 4.f, 3.2f, 0.5f}, {3.2f, 0.6f, 11.8f, 33.f}}
        };

        cv::Scalar val_init = params.at(CV_MAT_CN(OC)-1).init;
        cv::Scalar val_alpha = params.at(CV_MAT_CN(OC)-1).alpha;
        cv::Scalar val_sub = params.at(CV_MAT_CN(OC)-1).val_sub;
        cv::Scalar val_div = params.at(CV_MAT_CN(OC)-1).val_div;

        try {
            cv::cuda::GpuMat d_input(NUM_ELEMS_Y, NUM_ELEMS_X, I, val_init);
            cv::cuda::GpuMat d_crop = d_input(cv::Rect2d(cv::Point2d(200, 200), cv::Point2d(260, 320)));
            cv::Size up(64, 128);
            cv::cuda::GpuMat d_up(up, I);
            cv::cuda::GpuMat d_temp(up, OC);
            cv::cuda::GpuMat d_temp2(up, OC);

            cv::Mat diff(up, CV_MAT_DEPTH(OC));
            std::vector<cv::Mat> h_cvResults(CV_MAT_CN(OC));
            std::vector<cv::Mat> h_cvGSResults(CV_MAT_CN(OC));
            std::vector<cv::cuda::GpuMat> d_output_cv(CV_MAT_CN(OC));
            std::vector<cv::cuda::GpuMat> d_output_cvGS(CV_MAT_CN(OC));

            for (int i=0; i<CV_MAT_CN(I); i++) {
                d_output_cv.at(i).create(up, CV_MAT_DEPTH(OC));
                h_cvResults.at(i).create(up, CV_MAT_DEPTH(OC));
                d_output_cvGS.at(i).create(up, CV_MAT_DEPTH(OC));
                h_cvGSResults.at(i).create(up, CV_MAT_DEPTH(OC));
            }

            // OpenCV version
            cv::cuda::resize(d_crop, d_up, up, 0., 0., cv::INTER_LINEAR, cv_stream);
            d_up.convertTo(d_temp, OC, alpha, cv_stream);
            cv::cuda::subtract(d_temp, val_sub, d_temp2, cv::noArray(), -1, cv_stream);
            cv::cuda::divide(d_temp2, val_div, d_temp, 1.0, -1, cv_stream);
            cv::cuda::split(d_temp, d_output_cv, cv_stream);

            // cvGPUSpeedup version
            cvGS::executeOperations<I>(cv_stream,
                                       cvGS::resize<I, cv::INTER_LINEAR>(d_input, up, 0., 0.),
                                       cvGS::convertTo<I, OC>(),
                                       cvGS::multiply<OC>(val_alpha),
                                       cvGS::subtract<OC>(val_sub),
                                       cvGS::divide<OC>(val_div),
                                       cvGS::split<OC>(d_output_cvGS));

            // Looking at Nsight Systems, with an RTX A2000 12GB
            // Speedups are up to 7x, depending on the data type

            // Verify results
            for (int i=0; i<CV_MAT_CN(OC); i++) {
                d_output_cv.at(i).download(h_cvResults.at(i), cv_stream);
                d_output_cvGS.at(i).download(h_cvGSResults.at(i), cv_stream);
            }

            cv_stream.waitForCompletion();

            for (int i=0; i<CV_MAT_CN(OC); i++) {
                diff = cv::abs(h_cvResults.at(i) - h_cvGSResults.at(i));
                passed &= checkResults<CV_MAT_DEPTH(OC)>(diff.cols, diff.rows, diff);
            }
        } catch (const cv::Exception& e) {
            if (e.code != -210) {
                error_s << e.what();
                passed = false;
                exception = true;
            }
        } catch (const std::exception& e) {
            error_s << e.what();
            passed = false;
            exception = true;
        } 

        if (!passed) {
            if (!exception) {
                std::stringstream ss;
                ss << "testNoDefinedOutputOperation<" << cvTypeToString<I>() << ", " << cvTypeToString<OC>();
                std::cout << ss.str() << "> failed!! RESULT ERROR: Some results do not match baseline." << std::endl;
            } else {
                std::stringstream ss;
                ss << "testNoDefinedOutputOperation<" << cvTypeToString<I>() << ", " << cvTypeToString<OC>();
                std::cout << ss.str() << "> failed!! EXCEPTION: " << error_s.str() << std::endl;
            }
        }
    }

    return passed;
}

template <int I, int OC>
bool testNoDefinedOutputOperation(int NUM_ELEMS_X, int NUM_ELEMS_Y, cv::cuda::Stream& cv_stream, bool enabled) {
    std::stringstream error_s;
    bool passed = true;
    bool exception = false;

    if (enabled) {

        struct Parameters {
            cv::Scalar init;
            cv::Scalar val_sub;
            cv::Scalar val_mul;
            cv::Scalar val_div;
        };

        std::vector<Parameters> params = {
            {{2u}, {0.3f}, {1.f}, {3.2f}},
            {{2u, 37u}, {0.3f, 0.3f}, {1.f, 4.f}, {3.2f, 0.6f}},
            {{2u, 37u, 128u}, {0.3f, 0.3f, 0.3f}, {1.f, 4.f, 3.2f}, {3.2f, 0.6f, 11.8f}},
            {{2u, 37u, 128u, 20u}, {0.3f, 0.3f, 0.3f, 0.3f}, {1.f, 4.f, 3.2f, 0.5f}, {3.2f, 0.6f, 11.8f, 33.f}}
        };

        cv::Scalar val_init = params.at(CV_MAT_CN(OC)-1).init;
        cv::Scalar val_sub = params.at(CV_MAT_CN(OC)-1).val_sub;
        cv::Scalar val_mul = params.at(CV_MAT_CN(OC)-1).val_mul;
        cv::Scalar val_div = params.at(CV_MAT_CN(OC)-1).val_div;
        cv::Scalar val_add = params.at(CV_MAT_CN(OC)-1).val_div;

        try {
            cv::cuda::GpuMat d_input(NUM_ELEMS_Y, NUM_ELEMS_X, I, val_init);
            cv::cuda::GpuMat d_temp(NUM_ELEMS_Y, NUM_ELEMS_X, OC);
            cv::cuda::GpuMat d_output_cv(NUM_ELEMS_Y, NUM_ELEMS_X, OC);
            cv::cuda::GpuMat d_output_cvGS(NUM_ELEMS_Y, NUM_ELEMS_X, OC);

            cv::Mat h_cvResults(NUM_ELEMS_Y, NUM_ELEMS_X, OC);
            cv::Mat h_cvGSResults(NUM_ELEMS_Y, NUM_ELEMS_X, OC);

            // OpenCV version
            d_input.convertTo(d_temp, OC, cv_stream);
            cv::cuda::subtract(d_temp, val_sub, d_output_cv, cv::noArray(), -1, cv_stream);
            cv::cuda::multiply(d_output_cv, val_mul, d_temp, 1.0, -1, cv_stream);
            cv::cuda::divide(d_temp, val_div, d_output_cv, 1.0, -1, cv_stream);
            cv::cuda::add(d_output_cv, val_add, d_output_cv, cv::noArray(), -1, cv_stream);     

            // cvGPUSpeedup version
            cvGS::executeOperations<I, OC>(d_input, d_output_cvGS, cv_stream, 
                                            cvGS::convertTo<I, OC>(),
                                            cvGS::subtract<OC>(val_sub),
                                            cvGS::multiply<OC>(val_mul),
                                            cvGS::divide<OC>(val_div),
                                            cvGS::add<OC>(val_add));

            // Looking at Nsight Systems, with an RTX A2000 12GB
            // Speedups are up to 7x, depending on the data type

            // Verify results
            d_output_cv.download(h_cvResults, cv_stream);
            d_output_cvGS.download(h_cvGSResults, cv_stream);

            cv_stream.waitForCompletion();

            passed = compareAndCheck<OC>(NUM_ELEMS_X, NUM_ELEMS_Y, h_cvResults, h_cvGSResults);
            
        } catch (const std::exception& e) {
            error_s << e.what();
            passed = false;
            exception = true;
        }

        if (!passed) {
            if (!exception) {
                std::stringstream ss;
                ss << "testNoDefinedOutputOperation<" << cvTypeToString<I>() << ", " << cvTypeToString<OC>();
                std::cout << ss.str() << "> failed!! RESULT ERROR: Some results do not match baseline." << std::endl;
            } else {
                std::stringstream ss;
                ss << "testNoDefinedOutputOperation<" << cvTypeToString<I>() << ", " << cvTypeToString<OC>();
                std::cout << ss.str() << "> failed!! EXCEPTION: " << error_s.str() << std::endl;
            }
        }

    }

    return passed;
}

template <int I, int O>
bool testResize(int NUM_ELEMS_X, int NUM_ELEMS_Y, cv::cuda::Stream& cv_stream, bool enabled) {
    std::stringstream error_s;
    bool passed = true;
    bool exception = false;

    if (enabled) {

        struct Parameters {
            cv::Scalar init;
        };

        std::vector<Parameters> params = {
            {{2u}},
            {{2u, 37u}},
            {{2u, 37u, 128u}},
            {{2u, 37u, 128u, 20u}}
        };

        cv::Scalar val_init = params.at(CV_MAT_CN(I)-1).init;

        try {

            cv::cuda::GpuMat d_input(NUM_ELEMS_Y, NUM_ELEMS_X, I, val_init);

            cv::Size up(3870, 2260); // x,y
            cv::Size down(300, 500); // x,y

            cv::cuda::GpuMat d_down(down, I);
            cv::cuda::GpuMat d_up(up, I);

            cv::cuda::GpuMat d_down_cvGS(down, I);
            cv::cuda::GpuMat d_up_cvGS(up, I);

            cv::cuda::resize(d_input, d_up, up, 0., 0., cv::INTER_LINEAR, cv_stream);
            cv::cuda::resize(d_input, d_down, down, 0., 0., cv::INTER_LINEAR, cv_stream);

            cvGS::executeOperations<I>(cv_stream, cvGS::resize<I, cv::INTER_LINEAR>(d_input, up, 0., 0.), cvGS::write<I>(d_up_cvGS));
            cvGS::executeOperations<I>(cv_stream, cvGS::resize<I, cv::INTER_LINEAR>(d_input, down, 0., 0.), cvGS::write<I>(d_down_cvGS));
            cv::Mat h_up, h_up_cvGS;
            cv::Mat h_down, h_down_cvGS;

            d_up.download(h_up, cv_stream);
            d_up_cvGS.download(h_up_cvGS, cv_stream);
            d_down.download(h_down, cv_stream);
            d_down_cvGS.download(h_down_cvGS, cv_stream);

            cv_stream.waitForCompletion();

            passed &= compareAndCheck<I>(up.width, up.height, h_up, h_up_cvGS);
            passed &= compareAndCheck<I>(down.width, down.height, h_down, h_down_cvGS);

        } catch (const cv::Exception& e) {
            if (e.code != -210) {
                error_s << e.what();
                passed = false;
                exception = true;
            }
        } catch (const std::exception& e) {
            error_s << e.what();
            passed = false;
            exception = true;
        } 

        if (!passed) {
            if (!exception) {
                std::stringstream ss;
                ss << "testResize<" << cvTypeToString<I>() << ", " << cvTypeToString<O>();
                std::cout << ss.str() << "> failed!! RESULT ERROR: Some results do not match baseline." << std::endl;
            } else {
                std::stringstream ss;
                ss << "testResize<" << cvTypeToString<I>() << ", " << cvTypeToString<O>();
                std::cout << ss.str() << "> failed!! EXCEPTION: " << error_s.str() << std::endl;
            }
        }
    }

    return passed;
}

#define LAUNCH_TESTS(CV_INPUT, CV_OUTPUT) \
results["testNoDefinedOutputOperation"] &= testNoDefinedOutputOperation<CV_INPUT, CV_OUTPUT>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, true); \
results["testSplitOutputOperation"] &= testSplitOutputOperation<CV_INPUT, CV_OUTPUT>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, true); \
results["testResize"] &= testResize<CV_INPUT, CV_OUTPUT>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, true);

#define LAUNCH_TESTS_NO_SPLIT(CV_INPUT, CV_OUTPUT) \
results["testNoDefinedOutputOperation"] &= testNoDefinedOutputOperation<CV_INPUT, CV_OUTPUT>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, true); \
results["testResize"] &= testResize<CV_INPUT, CV_OUTPUT>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, true);

int main() {
    constexpr size_t NUM_ELEMS_X = 3840;
    constexpr size_t NUM_ELEMS_Y = 2160;

    cv::cuda::Stream cv_stream;

    cv::Mat::setDefaultAllocator(cv::cuda::HostMem::getAllocator(cv::cuda::HostMem::AllocType::PAGE_LOCKED));

    std::unordered_map<std::string, bool> results;
    results["testNoDefinedOutputOperation"] = true;
    results["testSplitOutputOperation"] = true;
    results["testResize"] = true;

    LAUNCH_TESTS_NO_SPLIT(CV_8UC1, CV_32FC1)
    LAUNCH_TESTS_NO_SPLIT(CV_8SC1, CV_32FC1)
    LAUNCH_TESTS_NO_SPLIT(CV_16UC1, CV_32FC1)
    LAUNCH_TESTS_NO_SPLIT(CV_16SC1, CV_32FC1)
    LAUNCH_TESTS_NO_SPLIT(CV_32SC1, CV_32FC1)
    LAUNCH_TESTS_NO_SPLIT(CV_32FC1, CV_32FC1)
    LAUNCH_TESTS(CV_8UC2, CV_32FC2)
    LAUNCH_TESTS(CV_8UC3, CV_32FC3)
    LAUNCH_TESTS(CV_8UC4, CV_32FC4)
    LAUNCH_TESTS(CV_8SC2, CV_32FC2)
    LAUNCH_TESTS(CV_8SC3, CV_32FC3)
    LAUNCH_TESTS(CV_8SC4, CV_32FC4)
    LAUNCH_TESTS(CV_16UC2, CV_32FC2)
    LAUNCH_TESTS(CV_16UC3, CV_32FC3)
    LAUNCH_TESTS(CV_16UC4, CV_32FC4)
    LAUNCH_TESTS(CV_16SC2, CV_32FC2)
    LAUNCH_TESTS(CV_16SC3, CV_32FC3)
    LAUNCH_TESTS(CV_16SC4, CV_32FC4)
    LAUNCH_TESTS(CV_32SC2, CV_32FC2)
    LAUNCH_TESTS(CV_32SC3, CV_32FC3)
    LAUNCH_TESTS(CV_32SC4, CV_32FC4)
    LAUNCH_TESTS(CV_32FC2, CV_64FC2)
    LAUNCH_TESTS(CV_32FC3, CV_64FC3)
    LAUNCH_TESTS(CV_32FC4, CV_64FC4)

    #undef LAUNCH_TESTS_NO_SPLIT
    #undef LAUNCH_TESTS

    for (const auto& [key, passed] : results) {
        if (passed) {
            std::cout << key << " passed!!" << std::endl;
        } else {
            std::cout << key << " failed!!" << std::endl;
        }
    }

    return 0;
}