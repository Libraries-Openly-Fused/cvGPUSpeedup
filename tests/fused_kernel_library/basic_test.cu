#include "hip/hip_runtime.h"
/* Copyright 2023-2025 Oscar Amoros Huguet

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <iostream>

#include <fused_kernel/core/data/ptr_nd.cuh>
#include <fused_kernel/core/execution_model/instantiable_operations.cuh>
#include <fused_kernel/core/execution_model/memory_operations.cuh>
#include <fused_kernel/algorithms/basic_ops/arithmetic.cuh>
#include <fused_kernel/algorithms/image_processing/saturate.cuh>
#include <fused_kernel/fused_kernel.cuh>
#include <fused_kernel/algorithms/basic_ops/arithmetic.cuh>
#include <fused_kernel/core/utils/template_operations.h>
#include <fused_kernel/algorithms/image_processing/saturate.cuh>

#include "tests/main.h"

template <typename T>
bool testPtr_2D() {
    constexpr size_t width = 1920;
    constexpr size_t height = 1080;
    constexpr size_t width_crop = 300;
    constexpr size_t height_crop = 200;

    fk::Point startPoint = {100, 200};

    fk::Ptr2D<T> input(width, height);
    fk::Ptr2D<T> cropedInput = input.crop(startPoint, fk::PtrDims<fk::_2D>(width_crop, height_crop));
    fk::Ptr2D<T> output(width_crop, height_crop);
    fk::Ptr2D<T> outputBig(width, height);

    hipStream_t stream;
    gpuErrchk(hipStreamCreate(&stream));

    fk::ReadInstantiableOperation<fk::PerThreadRead<fk::_2D, T>> readCrop{{cropedInput}};
    fk::ReadInstantiableOperation<fk::PerThreadRead<fk::_2D, T>> readFull{{input}};

    fk::WriteInstantiableOperation<fk::PerThreadWrite<fk::_2D, T>> opFinal_2D = { {output} };
    fk::WriteInstantiableOperation<fk::PerThreadWrite<fk::_2D, T>> opFinal_2DBig = { {outputBig} };

    for (int i=0; i<100; i++) {
        fk::executeOperations(stream, readCrop, opFinal_2D);
        fk::executeOperations(stream, readFull, opFinal_2DBig);
    }

    hipError_t err = hipStreamSynchronize(stream);

    // TODO: use some values and check results correctness

    if (err != hipSuccess) {
        return false;
    } else {
        return true;
    }
}

int launch() {
    bool test2Dpassed = true;

    test2Dpassed &= testPtr_2D<uchar>();
    test2Dpassed &= testPtr_2D<uchar3>();
    test2Dpassed &= testPtr_2D<float>();
    test2Dpassed &= testPtr_2D<float3>();

    hipStream_t stream;
    gpuErrchk(hipStreamCreate(&stream));

    fk::Ptr2D<uchar> input(64,64);
    fk::Ptr2D<uint> output(64,64);

    fk::Read<fk::PerThreadRead<fk::_2D, uchar>> read{ {input} };
    fk::Unary<fk::SaturateCast<uchar, uint>> cast = {};
    fk::Write<fk::PerThreadWrite<fk::_2D, uint>> write { {output} };

    auto fusedDF = fk::fuseDF(read, cast, fk::Binary<fk::Mul<uint>>{4});
    static_assert(std::is_same_v<std::decay_t<decltype(fusedDF.params.instance.params)>, fk::RawPtr<fk::_2D, uchar>>, "Unexpected type for params");
    //fusedDF.params.next.instance.params; // Should not compile
    static_assert(std::is_same_v<std::decay_t<decltype(fusedDF.params.next.next.instance.params)>, uint>, "Unexpected type for params");

    fk::executeOperations(stream, fusedDF, write);

    fk::OperationTuple<fk::PerThreadRead<fk::_2D, uchar>, fk::SaturateCast<uchar, uint>, fk::PerThreadWrite<fk::_2D, uint>> myTup{};

    fk::get<2>(myTup);
    constexpr bool test1 = std::is_same_v<fk::get_type_t<0, decltype(myTup)>, fk::PerThreadRead<fk::_2D, uchar>>;
    constexpr bool test2 = std::is_same_v<fk::get_type_t<1, decltype(myTup)>, fk::SaturateCast<uchar, uint>>;
    constexpr bool test3 = std::is_same_v<fk::get_type_t<2, decltype(myTup)>, fk::PerThreadWrite<fk::_2D, uint>>;

    gpuErrchk(hipStreamSynchronize(stream));

    if (test2Dpassed && fk::and_v<test1, test2, test3>) {
        std::cout << "cuda_transform executed!!" << std::endl;
        return 0;
    } else {
        std::cout << "cuda_transform failed!!" << std::endl;
        return -1;
    }
}