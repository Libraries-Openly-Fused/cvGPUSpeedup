#include "hip/hip_runtime.h"
/* Copyright 2023 Oscar Amoros Huguet

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <fstream>
#include <iostream>

#include "tests/testsCommon.cuh"
#include <opencv2/opencv.hpp>
#include <cvGPUSpeedup.cuh>

#include "tests/main.h"

#ifdef ENABLE_TEST_FUSED_RESIZE

struct PerPlaneSequenceSelector {
    FK_HOST_DEVICE_FUSE uint at(const uint& index) {
        return 1;
    }
};

void testComputeWhatYouSeePlusHorizontalFusion(char* buffer) {
    constexpr uint NUM_ELEMS_X = 3840;
    constexpr uint NUM_ELEMS_Y = 2160;

    hipStream_t stream;
    gpuErrchk(hipStreamCreate(&stream));

    constexpr fk::Size down(1920, 1080);
    cv::Mat h_result(down.height, down.width, CV_8UC4);
    cv::Mat nv12Image(cv::Size(NUM_ELEMS_X, NUM_ELEMS_Y + (NUM_ELEMS_Y / 2)), CV_8UC1, buffer);

    uchar* d_dataSource;
    size_t sourcePitch;
    gpuErrchk(hipMallocPitch(&d_dataSource, &sourcePitch, NUM_ELEMS_X, NUM_ELEMS_Y + (NUM_ELEMS_Y / 2)));
    fk::RawPtr<fk::_2D, uchar> d_nv12Image{ d_dataSource, {NUM_ELEMS_X, NUM_ELEMS_Y, (uint)sourcePitch} };
    fk::Ptr2D<uchar4> d_rgbaImage(down.width, down.height);
    fk::Ptr2D<uchar4> d_rgbaImageBig(NUM_ELEMS_X, NUM_ELEMS_Y);

    gpuErrchk(hipMemcpy2DAsync(d_nv12Image.data, d_nv12Image.dims.pitch,
        nv12Image.data, nv12Image.step,
        NUM_ELEMS_X, NUM_ELEMS_Y + (NUM_ELEMS_Y / 2), hipMemcpyHostToDevice, stream));
    constexpr int CAMERAS = 4;
    constexpr int OUTPUTS = 1;
    for (int i = 0; i < CAMERAS; i++) {
        fk::Read<fk::ReadYUV<fk::NV12>> read{ d_nv12Image, {NUM_ELEMS_X, NUM_ELEMS_Y} };
        fk::Unary<fk::ConvertYUVToRGB<fk::NV12, fk::Full, fk::bt601, true>> cvtColor{};
        fk::Write<fk::PerThreadWrite<fk::_2D, uchar4>> write{ d_rgbaImageBig.ptr() };
        fk::executeOperations(stream, read, cvtColor, write);

        fk::Read<fk::PerThreadRead<fk::_2D, uchar4>> read2{ d_rgbaImageBig.ptr(), {NUM_ELEMS_X, NUM_ELEMS_Y} };
        fk::Unary<fk::VectorReorder<uchar4, 2, 1, 0, 3>> cvtColor2{};
        fk::Write<fk::PerThreadWrite<fk::_2D, uchar4>> write2{ d_rgbaImageBig.ptr() };
        fk::executeOperations(stream, read2, cvtColor2, write2);
    }

    for (int i = 0; i < OUTPUTS; i++) {
        auto read3 = fk::resize<uchar4, fk::INTER_LINEAR>(d_rgbaImageBig.ptr(), down, 0., 0.);
        fk::Unary<fk::SaturateCast<float4, uchar4>> convertTo3{};
        fk::Write<fk::PerThreadWrite<fk::_2D, uchar4>> write3{ d_rgbaImage.ptr() };
        fk::executeOperations(stream, read3, convertTo3, write3);
    }

    gpuErrchk(hipMemcpy2DAsync(h_result.data, h_result.step,
        d_rgbaImage.ptr().data, d_rgbaImage.dims().pitch,
        down.width * sizeof(uchar4), down.height, hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipStreamSynchronize(stream));

    using PixelReadOp = fk::ComposedOperationSequence<fk::ReadYUV<fk::NV12>, fk::ConvertYUVToRGB<fk::NV12, fk::Full, fk::bt709, true, float4>>;
    fk::Binary<PixelReadOp> readOpInstance = { {{d_nv12Image}} };
    auto imgSize = d_nv12Image.dims;
    auto readOp = fk::resize<PixelReadOp, fk::INTER_LINEAR>(readOpInstance.params, fk::Size(NUM_ELEMS_X, NUM_ELEMS_Y), down);
    auto convertOp = fk::Unary<fk::SaturateCast<float4, uchar4>>{};
    auto colorConvert = fk::Unary<fk::VectorReorder<uchar4, 2, 1, 0, 3>>{};

    fk::Write<fk::TensorWrite<uchar4>> writesTensor;
    fk::Tensor<uchar4> myTensor(down.width, down.height, OUTPUTS);
    writesTensor.params = myTensor;

    auto OpSeqTensor = fk::buildOperationSequence(readOp, convertOp, colorConvert, writesTensor);

    dim3 block = myTensor.getBlockSize();
    dim3 grid((uint)ceil((float)down.width / (float)block.x),
              (uint)ceil((float)down.height / (float)block.y),
              (uint)OUTPUTS);

    fk::cuda_transform_divergent_batch<PerPlaneSequenceSelector><<<grid, block, 0, stream>>>(OpSeqTensor);
   
    gpuErrchk(hipStreamSynchronize(stream));

    gpuErrchk(hipFree(d_dataSource));

    gpuErrchk(hipStreamDestroy(stream));
}

void testComputeWhatYouSee(char* buffer) {
    constexpr uint NUM_ELEMS_X = 7680;
    constexpr uint NUM_ELEMS_Y = 4320;

    hipStream_t stream;
    gpuErrchk(hipStreamCreate(&stream));

    constexpr fk::Size down(1920, 1080);
    cv::Mat h_result(down.height, down.width, CV_8UC4);
    cv::Mat nv12Image(cv::Size(NUM_ELEMS_X, NUM_ELEMS_Y + (NUM_ELEMS_Y / 2)), CV_8UC1, buffer);

    uchar* d_dataSource;
    size_t sourcePitch;
    gpuErrchk(hipMallocPitch(&d_dataSource, &sourcePitch, NUM_ELEMS_X, NUM_ELEMS_Y + (NUM_ELEMS_Y / 2)));
    fk::RawPtr<fk::_2D, uchar> d_nv12Image{ d_dataSource, {NUM_ELEMS_X, NUM_ELEMS_Y, (uint)sourcePitch} };
    fk::Ptr2D<uchar4> d_rgbaImage(down.width, down.height);
    fk::Ptr2D<uchar4> d_rgbaImageBig(NUM_ELEMS_X, NUM_ELEMS_Y);

    gpuErrchk(hipMemcpy2DAsync(d_nv12Image.data, d_nv12Image.dims.pitch,
        nv12Image.data, nv12Image.step,
        NUM_ELEMS_X, NUM_ELEMS_Y + (NUM_ELEMS_Y / 2), hipMemcpyHostToDevice, stream));

    fk::Read<fk::ReadYUV<fk::NV12>> read{ d_nv12Image, {NUM_ELEMS_X, NUM_ELEMS_Y} };
    fk::Unary<fk::ConvertYUVToRGB<fk::NV12, fk::Full, fk::bt601, true>> cvtColor{};
    fk::Write<fk::PerThreadWrite<fk::_2D, uchar4>> write{ d_rgbaImageBig.ptr() };
    fk::executeOperations(stream, read, cvtColor, write);

    fk::Read<fk::PerThreadRead<fk::_2D, uchar4>> read2{ d_rgbaImageBig.ptr(), {NUM_ELEMS_X, NUM_ELEMS_Y} };
    fk::Unary<fk::VectorReorder<uchar4, 2, 1, 0, 3>> cvtColor2{};
    fk::Write<fk::PerThreadWrite<fk::_2D, uchar4>> write2{ d_rgbaImageBig.ptr() };
    fk::executeOperations(stream, read2, cvtColor2, write2);

    auto read3 = fk::resize<uchar4, fk::INTER_LINEAR>(d_rgbaImageBig.ptr(), down, 0., 0.);
    fk::Unary<fk::SaturateCast<float4, uchar4>> convertTo3{};
    fk::Write<fk::PerThreadWrite<fk::_2D, uchar4>> write3{ d_rgbaImage.ptr() };
    fk::executeOperations(stream, read3, convertTo3, write3);

    gpuErrchk(hipMemcpy2DAsync(h_result.data, h_result.step,
        d_rgbaImage.ptr().data, d_rgbaImage.dims().pitch,
        down.width * sizeof(uchar4), down.height, hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipStreamSynchronize(stream));

    using PixelReadOp = fk::ComposedOperationSequence<fk::ReadYUV<fk::NV12>, fk::ConvertYUVToRGB<fk::NV12, fk::Full, fk::bt709, true, float4>>;
    fk::Binary<PixelReadOp> readOpInstance = { {{d_nv12Image}} };
    auto imgSize = d_nv12Image.dims;
    auto readOp = fk::resize<PixelReadOp, fk::INTER_LINEAR>(readOpInstance.params, fk::Size(imgSize.width, imgSize.height), down);
    auto convertOp = fk::Unary<fk::SaturateCast<float4, uchar4>>{};
    auto colorConvert = fk::Unary<fk::VectorReorder<uchar4, 2, 1, 0, 3>>{};
    auto writeOp = fk::Write<fk::PerThreadWrite<fk::_2D, uchar4>>{ d_rgbaImage.ptr() };
    fk::executeOperations(stream, readOp, convertOp, colorConvert, writeOp);
    gpuErrchk(hipMemcpy2DAsync(h_result.data, h_result.step,
        d_rgbaImage.ptr().data, d_rgbaImage.dims().pitch,
        down.width * sizeof(uchar4), down.height, hipMemcpyDeviceToHost, stream));

    gpuErrchk(hipStreamSynchronize(stream));

    gpuErrchk(hipFree(d_dataSource));

    gpuErrchk(hipStreamDestroy(stream));
}
#endif

int launch() {
    int returnValue = 0;
#ifdef ENABLE_TEST_FUSED_RESIZE
    cv::cuda::Stream cv_stream;

    cv::Mat::setDefaultAllocator(cv::cuda::HostMem::getAllocator(cv::cuda::HostMem::AllocType::PAGE_LOCKED));

    const std::string filePath{ "" };
    std::ifstream file(filePath, std::ios::binary | std::ios::ate);
    std::streamsize size = file.tellg();
    file.seekg(0, std::ios::beg);

    if (file.is_open()) {
        char* buffer = new char[size];
        file.read(buffer, size);

        testComputeWhatYouSee(buffer);
        delete buffer;
    } else {
        // Print an error message if the file cannot be opened
        std::cerr << "Error: cannot open file\n";
        returnValue = -1;
    }
    file.close();

    const std::string filePath2{ "" };
    std::ifstream file2(filePath2, std::ios::binary | std::ios::ate);
    std::streamsize size2 = file2.tellg();
    file2.seekg(0, std::ios::beg);

    if (file2.is_open()) {
        char* buffer = new char[size2];
        file2.read(buffer, size2);

        testComputeWhatYouSeePlusHorizontalFusion(buffer);
        delete buffer;
    } else {
        // Print an error message if the file cannot be opened
        std::cerr << "Error: cannot open file\n";
        returnValue = -1;
    }
    file2.close();
#endif
    return returnValue;
}