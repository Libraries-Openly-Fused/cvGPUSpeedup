#include "hip/hip_runtime.h"
/* Copyright 2023 Oscar Amoros Huguet

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <sstream>

#include "testsCommon.cuh"
#include <cvGPUSpeedup.cuh>
#include <opencv2/cudaimgproc.hpp>

template <int CV_TYPE_I, int CV_TYPE_O, int CROPS>
bool test_batchresize_x_split3D_OCVBatch(int NUM_ELEMS_X, int NUM_ELEMS_Y, cv::cuda::Stream& cv_stream, bool enabled) {
    std::stringstream error_s;
    bool passed = true;
    bool exception = false;

    if (enabled) {
        struct Parameters {
            cv::Scalar init;
            cv::Scalar alpha;
            cv::Scalar val_sub;
            cv::Scalar val_div;
        };

        double alpha = 0.3;

        std::vector<Parameters> params = {
            {{2u}, {alpha}, {1.f}, {3.2f}},
            {{2u, 37u}, {alpha, alpha}, {1.f, 4.f}, {3.2f, 0.6f}},
            {{5u, 5u, 5u}, {alpha, alpha, alpha}, {1.f, 4.f, 3.2f}, {3.2f, 0.6f, 11.8f}},
            {{2u, 37u, 128u, 20u}, {alpha, alpha, alpha, alpha}, {1.f, 4.f, 3.2f, 0.5f}, {3.2f, 0.6f, 11.8f, 33.f}}
        };

        cv::Scalar val_init = params.at(CV_MAT_CN(CV_TYPE_O)-1).init;
        cv::Scalar val_alpha = params.at(CV_MAT_CN(CV_TYPE_O)-1).alpha;
        cv::Scalar val_sub = params.at(CV_MAT_CN(CV_TYPE_O)-1).val_sub;
        cv::Scalar val_div = params.at(CV_MAT_CN(CV_TYPE_O)-1).val_div;

        try {
            cv::cuda::GpuMat d_input(NUM_ELEMS_Y, NUM_ELEMS_X, CV_TYPE_I, val_init);
            std::array<cv::Rect2d, CROPS> crops_2d;
            for (int crop_i = 0; crop_i<CROPS; crop_i++) {
                crops_2d[crop_i] = cv::Rect2d(cv::Point2d(crop_i, crop_i), cv::Point2d(crop_i+60, crop_i+120));
            }

            cv::Size up(64, 128);
            cv::cuda::GpuMat d_up(up, CV_TYPE_I);

            std::array<std::vector<cv::Mat>, CROPS> h_cvResults;
            std::array<std::vector<cv::Mat>, CROPS> h_cvGSResults;
            cv::cuda::GpuMat d_tensor_output(CROPS, 
                                             up.width * up.height * CV_MAT_CN(CV_TYPE_O),
                                             CV_MAT_DEPTH(CV_TYPE_O),
                                             up.width * up.height * CV_MAT_CN(CV_TYPE_O) * sizeof(BASE_CUDA_T(CV_TYPE_O)));
            cv::cuda::GpuMat d_resize_output(CROPS, 
                                             up.width * up.height,
                                             CV_TYPE_I,
                                             up.width * up.height * CV_MAT_CN(CV_TYPE_I) * sizeof(BASE_CUDA_T(CV_TYPE_I)));
            std::array<cv::cuda::GpuMat, CROPS> d_resized_array;
            cv::cuda::GpuMat d_temp(CROPS, 
                                    up.width * up.height,
                                    CV_TYPE_O,
                                    up.width * up.height * CV_MAT_CN(CV_TYPE_O) * sizeof(BASE_CUDA_T(CV_TYPE_O)));
            cv::cuda::GpuMat d_temp2(CROPS,
                                     up.width * up.height,
                                     CV_TYPE_O,
                                     up.width * up.height * CV_MAT_CN(CV_TYPE_O) * sizeof(BASE_CUDA_T(CV_TYPE_O)));
            std::array<std::vector<cv::cuda::GpuMat>, CROPS> d_output_cv;

            cv::Mat diff(up, CV_MAT_DEPTH(CV_TYPE_O));
            cv::Mat h_tensor_output(CROPS, up.width * up.height * CV_MAT_CN(CV_TYPE_O), CV_MAT_DEPTH(CV_TYPE_O));

            std::array<cv::cuda::GpuMat, CROPS> crops;
            for (int crop_i=0; crop_i<CROPS; crop_i++) {
                crops[crop_i] = d_input(crops_2d[crop_i]);
                d_resized_array[crop_i] = cv::cuda::GpuMat(up, d_resize_output.type(), d_resize_output.row(crop_i).data);
                for (int i=0; i<CV_MAT_CN(CV_TYPE_I); i++) {
                    d_output_cv.at(crop_i).emplace_back(up, CV_MAT_DEPTH(CV_TYPE_O));
                    h_cvResults.at(crop_i).emplace_back(up, CV_MAT_DEPTH(CV_TYPE_O));
                }
            }

            constexpr bool correctDept = CV_MAT_DEPTH(CV_TYPE_O) == CV_32F;

            // OpenCV version
            for (int crop_i=0; crop_i<CROPS; crop_i++) {
                cv::cuda::resize(crops[crop_i], d_resized_array[crop_i], up, 0., 0., cv::INTER_LINEAR, cv_stream);
            }
            d_resize_output.convertTo(d_temp, CV_TYPE_O, alpha, cv_stream);

            if constexpr (CV_MAT_CN(CV_TYPE_O) == 3 && correctDept) {
                cv::cuda::cvtColor(d_temp, d_temp, cv::COLOR_RGB2BGR, 0, cv_stream);
            } else if constexpr (CV_MAT_CN(CV_TYPE_O) == 4 && correctDept) {
                cv::cuda::cvtColor(d_temp, d_temp, cv::COLOR_RGBA2BGRA, 0, cv_stream);
            }

            cv::cuda::subtract(d_temp, val_sub, d_temp2, cv::noArray(), -1, cv_stream);
            cv::cuda::divide(d_temp2, val_div, d_temp, 1.0, -1, cv_stream);
            for (int crop_i=0; crop_i<CROPS; crop_i++) {
                cv::cuda::split(d_temp.row(crop_i).reshape(CV_MAT_CN(CV_TYPE_O), up.height), d_output_cv[crop_i], cv_stream);
            }

            // cvGPUSpeedup version
            if constexpr (CV_MAT_CN(CV_TYPE_O) == 3 && correctDept) {
                cvGS::executeOperations(cv_stream,
                                        cvGS::resize<CV_TYPE_I, cv::INTER_LINEAR, CROPS, false>(crops, up, CROPS, fk::make_set<CUDA_T(CV_TYPE_I)>(0u)),
                                        cvGS::convertTo<CV_TYPE_I, CV_TYPE_O>(),
                                        cvGS::cvtColor<CV_TYPE_O, cv::COLOR_RGB2BGR>(),
                                        cvGS::multiply<CV_TYPE_O>(val_alpha),
                                        cvGS::subtract<CV_TYPE_O>(val_sub),
                                        cvGS::divide<CV_TYPE_O>(val_div),
                                        cvGS::split<CV_TYPE_O>(d_tensor_output, up));
            } else if constexpr (CV_MAT_CN(CV_TYPE_O) == 4 && correctDept) {
                cvGS::executeOperations(cv_stream,
                                        cvGS::resize<CV_TYPE_I, cv::INTER_LINEAR, CROPS, false>(crops, up, CROPS, fk::make_set<CUDA_T(CV_TYPE_I)>(0u)),
                                        cvGS::convertTo<CV_TYPE_I, CV_TYPE_O>(),
                                        cvGS::cvtColor<CV_TYPE_O, cv::COLOR_RGBA2BGRA>(),
                                        cvGS::multiply<CV_TYPE_O>(val_alpha),
                                        cvGS::subtract<CV_TYPE_O>(val_sub),
                                        cvGS::divide<CV_TYPE_O>(val_div),
                                        cvGS::split<CV_TYPE_O>(d_tensor_output, up));
            } else {
                cvGS::executeOperations(cv_stream,
                                        cvGS::resize<CV_TYPE_I, cv::INTER_LINEAR, CROPS, false>(crops, up, CROPS, fk::make_set<CUDA_T(CV_TYPE_I)>(0u)),
                                        cvGS::convertTo<CV_TYPE_I, CV_TYPE_O>(),
                                        cvGS::multiply<CV_TYPE_O>(val_alpha),
                                        cvGS::subtract<CV_TYPE_O>(val_sub),
                                        cvGS::divide<CV_TYPE_O>(val_div),
                                        cvGS::split<CV_TYPE_O>(d_tensor_output, up));
            }

            d_tensor_output.download(h_tensor_output, cv_stream);

            // Verify results
            for (int crop_i=0; crop_i<CROPS; crop_i++) {
                for (int i=0; i<CV_MAT_CN(CV_TYPE_O); i++) {
                    d_output_cv[crop_i].at(i).download(h_cvResults[crop_i].at(i), cv_stream);
                }
            }

            cv_stream.waitForCompletion();

            for (int crop_i=0; crop_i<CROPS; crop_i++) {
                cv::Mat row = h_tensor_output.row(crop_i);
                for (int i=0; i<CV_MAT_CN(CV_TYPE_O); i++) {
                    int planeStart = i * up.width*up.height;
                    int planeEnd = ((i+1) * up.width*up.height) - 1;
                    cv::Mat plane = row.colRange(planeStart, planeEnd);
                    h_cvGSResults[crop_i].push_back(cv::Mat(up.height, up.width, plane.type(), plane.data));
                }
            }

            for (int crop_i=0; crop_i<CROPS; crop_i++) {
                for (int i=0; i<CV_MAT_CN(CV_TYPE_O); i++) {
                    cv::Mat cvRes = h_cvResults[crop_i].at(i);
                    cv::Mat cvGSRes = h_cvGSResults[crop_i].at(i);
                    diff = cv::abs(cvRes - cvGSRes);
                    bool passedThisTime = checkResults<CV_MAT_DEPTH(CV_TYPE_O)>(diff.cols, diff.rows, diff);
                    passed &= passedThisTime;
                }
            }
        } catch (const cv::Exception& e) {
            if (e.code != -210) {
                error_s << e.what();
                passed = false;
                exception = true;
            }
        } catch (const std::exception& e) {
            error_s << e.what();
            passed = false;
            exception = true;
        } 

        if (!passed) {
            if (!exception) {
                std::stringstream ss;
                ss << "test_batchresize_x_split3D_OCVBatch<" << cvTypeToString<CV_TYPE_I>() << ", " << cvTypeToString<CV_TYPE_O>();
                std::cout << ss.str() << "> failed!! RESULT ERROR: Some results do not match baseline." << std::endl;
            } else {
                std::stringstream ss;
                ss << "test_batchresize_x_split3D_OCVBatch<" << cvTypeToString<CV_TYPE_I>() << ", " << cvTypeToString<CV_TYPE_O>();
                std::cout << ss.str() << "> failed!! EXCEPTION: " << error_s.str() << std::endl;
            }
        }
    }

    return passed;
}

template <int CV_TYPE_I, int CV_TYPE_O>
bool test_batchresize_x_split3D_OCVBatch_10_30_50_100(int NUM_ELEMS_X, int NUM_ELEMS_Y, cv::cuda::Stream& cv_stream, bool enabled) {
    bool passed = true;

    passed &= test_batchresize_x_split3D_OCVBatch<CV_TYPE_I, CV_TYPE_O, 10>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, enabled);
    passed &= test_batchresize_x_split3D_OCVBatch<CV_TYPE_I, CV_TYPE_O, 30>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, enabled);
    passed &= test_batchresize_x_split3D_OCVBatch<CV_TYPE_I, CV_TYPE_O, 50>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, enabled);
    passed &= test_batchresize_x_split3D_OCVBatch<CV_TYPE_I, CV_TYPE_O, 100>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, enabled);

    return passed;
}

template <int CV_TYPE_I, int CV_TYPE_O, int CROPS>
bool test_batchresize_x_split3D(int NUM_ELEMS_X, int NUM_ELEMS_Y, cv::cuda::Stream& cv_stream, bool enabled) {
    std::stringstream error_s;
    bool passed = true;
    bool exception = false;

    if (enabled) {
        struct Parameters {
            cv::Scalar init;
            cv::Scalar alpha;
            cv::Scalar val_sub;
            cv::Scalar val_div;
        };

        double alpha = 0.3;

        std::vector<Parameters> params = {
            {{2u}, {alpha}, {1.f}, {3.2f}},
            {{2u, 37u}, {alpha, alpha}, {1.f, 4.f}, {3.2f, 0.6f}},
            {{5u, 5u, 5u}, {alpha, alpha, alpha}, {1.f, 4.f, 3.2f}, {3.2f, 0.6f, 11.8f}},
            {{2u, 37u, 128u, 20u}, {alpha, alpha, alpha, alpha}, {1.f, 4.f, 3.2f, 0.5f}, {3.2f, 0.6f, 11.8f, 33.f}}
        };

        cv::Scalar val_init = params.at(CV_MAT_CN(CV_TYPE_O)-1).init;
        cv::Scalar val_alpha = params.at(CV_MAT_CN(CV_TYPE_O)-1).alpha;
        cv::Scalar val_sub = params.at(CV_MAT_CN(CV_TYPE_O)-1).val_sub;
        cv::Scalar val_div = params.at(CV_MAT_CN(CV_TYPE_O)-1).val_div;

        try {
            cv::cuda::GpuMat d_input(NUM_ELEMS_Y, NUM_ELEMS_X, CV_TYPE_I, val_init);
            std::array<cv::Rect2d, CROPS> crops_2d;
            for (int crop_i = 0; crop_i<CROPS; crop_i++) {
                crops_2d[crop_i] = cv::Rect2d(cv::Point2d(crop_i, crop_i), cv::Point2d(crop_i+60, crop_i+120));
            }

            cv::Size up(64, 128);
            cv::cuda::GpuMat d_up(up, CV_TYPE_I);
            cv::cuda::GpuMat d_temp(up, CV_TYPE_O);
            cv::cuda::GpuMat d_temp2(up, CV_TYPE_O);

            std::array<std::vector<cv::cuda::GpuMat>, CROPS> d_output_cv;
            std::array<std::vector<cv::cuda::GpuMat>, CROPS> d_output_cvGS;
            std::array<std::vector<cv::Mat>, CROPS> h_cvResults;
            std::array<std::vector<cv::Mat>, CROPS> h_cvGSResults;
            cv::cuda::GpuMat d_tensor_output(CROPS, 
                                             up.width * up.height * CV_MAT_CN(CV_TYPE_O),
                                             CV_MAT_DEPTH(CV_TYPE_O),
                                             up.width * up.height * CV_MAT_CN(CV_TYPE_O) * sizeof(BASE_CUDA_T(CV_TYPE_O)));

            cv::Mat diff(up, CV_MAT_DEPTH(CV_TYPE_O));
            cv::Mat h_tensor_output(CROPS, up.width * up.height * CV_MAT_CN(CV_TYPE_O), CV_MAT_DEPTH(CV_TYPE_O));

            std::array<cv::cuda::GpuMat, CROPS> crops;
            for (int crop_i=0; crop_i<CROPS; crop_i++) {
                crops[crop_i] = d_input(crops_2d[crop_i]);
                for (int i=0; i<CV_MAT_CN(CV_TYPE_I); i++) {
                    d_output_cv.at(crop_i).emplace_back(up, CV_MAT_DEPTH(CV_TYPE_O));
                    h_cvResults.at(crop_i).emplace_back(up, CV_MAT_DEPTH(CV_TYPE_O));
                }
            }

            constexpr bool correctDept = CV_MAT_DEPTH(CV_TYPE_O) == CV_32F;

            // OpenCV version
            for (int crop_i=0; crop_i<CROPS; crop_i++) {
                cv::cuda::resize(crops[crop_i], d_up, up, 0., 0., cv::INTER_LINEAR, cv_stream);
                d_up.convertTo(d_temp, CV_TYPE_O, alpha, cv_stream);
                if constexpr (CV_MAT_CN(CV_TYPE_I) == 3 && correctDept) {
                    cv::cuda::cvtColor(d_temp, d_temp, cv::COLOR_RGB2BGR, 0, cv_stream);
                } else if constexpr (CV_MAT_CN(CV_TYPE_I) == 4 && correctDept) {
                    cv::cuda::cvtColor(d_temp, d_temp, cv::COLOR_RGBA2BGRA, 0, cv_stream);
                }
                cv::cuda::subtract(d_temp, val_sub, d_temp2, cv::noArray(), -1, cv_stream);
                cv::cuda::divide(d_temp2, val_div, d_temp, 1.0, -1, cv_stream);
                cv::cuda::split(d_temp, d_output_cv[crop_i], cv_stream);
            }

            // cvGPUSpeedup
            if constexpr (CV_MAT_CN(CV_TYPE_I) == 3 && correctDept) {
                cvGS::executeOperations(cv_stream,
                                        cvGS::resize<CV_TYPE_I, cv::INTER_LINEAR, CROPS, false>(crops, up, CROPS, fk::make_set<CUDA_T(CV_TYPE_I)>(0u)),
                                        cvGS::convertTo<CV_TYPE_I, CV_TYPE_O>(),
                                        cvGS::cvtColor<CV_TYPE_O, cv::COLOR_RGB2BGR>(),
                                        cvGS::multiply<CV_TYPE_O>(val_alpha),
                                        cvGS::subtract<CV_TYPE_O>(val_sub),
                                        cvGS::divide<CV_TYPE_O>(val_div),
                                        cvGS::split<CV_TYPE_O>(d_tensor_output, up));
            } else if constexpr (CV_MAT_CN(CV_TYPE_I) == 4 && correctDept) {
                cvGS::executeOperations(cv_stream,
                                       cvGS::resize<CV_TYPE_I, cv::INTER_LINEAR, CROPS, false>(crops, up, CROPS, fk::make_set<CUDA_T(CV_TYPE_I)>(0u)),
                                       cvGS::convertTo<CV_TYPE_I, CV_TYPE_O>(),
                                       cvGS::cvtColor<CV_TYPE_O, cv::COLOR_RGBA2BGRA>(),
                                       cvGS::multiply<CV_TYPE_O>(val_alpha),
                                       cvGS::subtract<CV_TYPE_O>(val_sub),
                                       cvGS::divide<CV_TYPE_O>(val_div),
                                       cvGS::split<CV_TYPE_O>(d_tensor_output, up));
            } else {
                cvGS::executeOperations(cv_stream,
                                       cvGS::resize<CV_TYPE_I, cv::INTER_LINEAR, CROPS, false>(crops, up, CROPS, fk::make_set<CUDA_T(CV_TYPE_I)>(0u)),
                                       cvGS::convertTo<CV_TYPE_I, CV_TYPE_O>(),
                                       cvGS::multiply<CV_TYPE_O>(val_alpha),
                                       cvGS::subtract<CV_TYPE_O>(val_sub),
                                       cvGS::divide<CV_TYPE_O>(val_div),
                                       cvGS::split<CV_TYPE_O>(d_tensor_output, up));
            }

            d_tensor_output.download(h_tensor_output, cv_stream);

            // Verify results
            for (int crop_i=0; crop_i<CROPS; crop_i++) {
                for (int i=0; i<CV_MAT_CN(CV_TYPE_O); i++) {
                    d_output_cv[crop_i].at(i).download(h_cvResults[crop_i].at(i), cv_stream);
                }
            }

            cv_stream.waitForCompletion();

            for (int crop_i=0; crop_i<CROPS; crop_i++) {
                cv::Mat row = h_tensor_output.row(crop_i);
                for (int i=0; i<CV_MAT_CN(CV_TYPE_O); i++) {
                    int planeStart = i * up.width*up.height;
                    int planeEnd = ((i+1) * up.width*up.height) - 1;
                    cv::Mat plane = row.colRange(planeStart, planeEnd);
                    h_cvGSResults[crop_i].push_back(cv::Mat(up.height, up.width, plane.type(), plane.data));
                }
            }

            for (int crop_i=0; crop_i<CROPS; crop_i++) {
                for (int i=0; i<CV_MAT_CN(CV_TYPE_O); i++) {
                    cv::Mat cvRes = h_cvResults[crop_i].at(i);
                    cv::Mat cvGSRes = h_cvGSResults[crop_i].at(i);
                    diff = cv::abs(cvRes - cvGSRes);
                    bool passedThisTime = checkResults<CV_MAT_DEPTH(CV_TYPE_O)>(diff.cols, diff.rows, diff);
                    passed &= passedThisTime;
                }
            }
        } catch (const cv::Exception& e) {
            if (e.code != -210) {
                error_s << e.what();
                passed = false;
                exception = true;
            }
        } catch (const std::exception& e) {
            error_s << e.what();
            passed = false;
            exception = true;
        } 

        if (!passed) {
            if (!exception) {
                std::stringstream ss;
                ss << "test_batchresize_x_split3D<" << cvTypeToString<CV_TYPE_I>() << ", " << cvTypeToString<CV_TYPE_O>();
                std::cout << ss.str() << "> failed!! RESULT ERROR: Some results do not match baseline." << std::endl;
            } else {
                std::stringstream ss;
                ss << "test_batchresize_x_split3D<" << cvTypeToString<CV_TYPE_I>() << ", " << cvTypeToString<CV_TYPE_O>();
                std::cout << ss.str() << "> failed!! EXCEPTION: " << error_s.str() << std::endl;
            }
        }
    }

    return passed;
}

template <int CV_TYPE_I, int CV_TYPE_O>
bool test_batchresize_x_split3D_10_30_50_100(int NUM_ELEMS_X, int NUM_ELEMS_Y, cv::cuda::Stream& cv_stream, bool enabled) {
    bool passed = true;

    passed &= test_batchresize_x_split3D<CV_TYPE_I, CV_TYPE_O, 10>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, enabled);
    passed &= test_batchresize_x_split3D<CV_TYPE_I, CV_TYPE_O, 30>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, enabled);
    passed &= test_batchresize_x_split3D<CV_TYPE_I, CV_TYPE_O, 50>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, enabled);
    passed &= test_batchresize_x_split3D<CV_TYPE_I, CV_TYPE_O, 100>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, enabled);

    return passed;
}

int main() {
    constexpr size_t NUM_ELEMS_X = 3840;
    constexpr size_t NUM_ELEMS_Y = 2160;

    cv::cuda::Stream cv_stream;

    cv::Mat::setDefaultAllocator(cv::cuda::HostMem::getAllocator(cv::cuda::HostMem::AllocType::PAGE_LOCKED));

    std::unordered_map<std::string, bool> results;
    results["test_batchresize_x_split3D_OCVBatch_10_30_50_100"] = true;
    results["test_batchresize_x_split3D_10_30_50_100"] = true;

    #define LAUNCH_TESTS(CV_INPUT, CV_OUTPUT) \
    results["test_batchresize_x_split3D_OCVBatch_10_30_50_100"] &= test_batchresize_x_split3D_OCVBatch_10_30_50_100<CV_INPUT, CV_OUTPUT>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, true); \
    results["test_batchresize_x_split3D_10_30_50_100"] &= test_batchresize_x_split3D_10_30_50_100<CV_INPUT, CV_OUTPUT>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, true);

    LAUNCH_TESTS(CV_8UC3, CV_32FC3)
    LAUNCH_TESTS(CV_8UC4, CV_32FC4)
    LAUNCH_TESTS(CV_16UC3, CV_32FC3)
    LAUNCH_TESTS(CV_16UC4, CV_32FC4)
    LAUNCH_TESTS(CV_16SC3, CV_32FC3)
    LAUNCH_TESTS(CV_16SC4, CV_32FC4)
    LAUNCH_TESTS(CV_32FC3, CV_64FC3)
    LAUNCH_TESTS(CV_32FC4, CV_64FC4)

    #undef LAUNCH_TESTS

    for (const auto& [key, passed] : results) {
        if (passed) {
            std::cout << key << " passed!!" << std::endl;
        } else {
            std::cout << key << " failed!!" << std::endl;
        }
    }

    return 0;
}