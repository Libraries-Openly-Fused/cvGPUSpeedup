#include "hip/hip_runtime.h"
/* Copyright 2025 Grup Mediapro S.L.U

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "tests/main.h"

#include <cvGPUSpeedup.cuh>
#include <opencv2/opencv.hpp>

bool testPerspective() {
    // Load the image
    const cv::Mat img = cv::imread("E:/GitHub/cvGPUSpeedup/images/NSightSystemsTimeline1.png");
    if (img.empty()) {
        std::cerr << "Error loading image" << std::endl;
        return -1;
    }

    cv::cuda::Stream stream;

    // Upload the image to GPU
    const cv::cuda::GpuMat d_img(img);

    // Define the source and destination points for perspective transformation
    cv::Point2f src_points[4] = { cv::Point2f(56, 65), cv::Point2f(368, 52), cv::Point2f(28, 387), cv::Point2f(389, 390) };
    cv::Point2f dst_points[4] = { cv::Point2f(0, 0), cv::Point2f(300, 0), cv::Point2f(0, 300), cv::Point2f(300, 300) };

    // Get the perspective transformation matrix
    cv::Mat perspective_matrix = cv::getPerspectiveTransform(src_points, dst_points);

    // Preallocate the result images
    cv::cuda::GpuMat d_resultcv(img.size(), CV_8UC3);
    cv::cuda::GpuMat d_resultcvGS(img.size(), CV_8UC3);

    // Apply the perspective transformation
    cv::cuda::warpPerspective(d_img, d_resultcv, perspective_matrix, img.size(), 1, 0, cv::Scalar(), stream);

    cv::Mat inverted_perspective_matrix;
    invert(perspective_matrix, inverted_perspective_matrix);

    const auto warpFunc = cvGS::warp<fk::WarpType::Perspective, CV_8UC3>(d_img, inverted_perspective_matrix, img.size());

    bool correct{ true };
    /*const double* const rawMat = perspective_matrix.ptr<double>();
    correct &= std::abs(static_cast<float>(rawMat[0]) - warpFunc.params.transformMatrix.data[0][0]) < 0.001;
    correct &= std::abs(static_cast<float>(rawMat[1]) - warpFunc.params.transformMatrix.data[0][1]) < 0.001;
    correct &= std::abs(static_cast<float>(rawMat[2]) - warpFunc.params.transformMatrix.data[0][2]) < 0.001;
    correct &= std::abs(static_cast<float>(rawMat[3]) - warpFunc.params.transformMatrix.data[1][0]) < 0.001;
    correct &= std::abs(static_cast<float>(rawMat[4]) - warpFunc.params.transformMatrix.data[1][1]) < 0.001;
    correct &= std::abs(static_cast<float>(rawMat[5]) - warpFunc.params.transformMatrix.data[1][2]) < 0.001;
    correct &= std::abs(static_cast<float>(rawMat[6]) - warpFunc.params.transformMatrix.data[2][0]) < 0.001;
    correct &= std::abs(static_cast<float>(rawMat[7]) - warpFunc.params.transformMatrix.data[2][1]) < 0.001;
    correct &= std::abs(static_cast<float>(rawMat[8]) - warpFunc.params.transformMatrix.data[2][2]) < 0.001;*/

    auto writeFunc = cvGS::write<CV_8UC3>(d_resultcvGS);
    cvGS::executeOperations(stream, warpFunc, fk::Cast<float3, uchar3>::build(), writeFunc);

    stream.waitForCompletion();

    // Download the result back to CPU
    cv::Mat resultcv(d_resultcv);
    cv::Mat resultcvGS(d_resultcvGS);

    return correct;
}

bool testAffine() {
    // Load the image
    const cv::Mat img = cv::imread("E:/GitHub/cvGPUSpeedup/images/NSightSystemsTimeline1.png");
    if (img.empty()) {
        std::cerr << "Error loading image" << std::endl;
        return -1;
    }

    cv::cuda::Stream stream;

    // Upload the image to GPU
    const cv::cuda::GpuMat d_img(img);

    // Define the translation values
    double tx = 50, ty = 100;

    // Get the affine transformation matrix
    cv::Mat affine_matrix = (cv::Mat_<double>(2, 3) << 1, 0, tx, 0, 1, ty);

    // Preallocate the result images
    cv::cuda::GpuMat d_resultcv(img.size(), CV_8UC3);
    cv::cuda::GpuMat d_resultcvGS(img.size(), CV_8UC3);

    // Apply the affine transformation
    cv::cuda::GpuMat d_result;
    cv::cuda::warpAffine(d_img, d_resultcv, affine_matrix, img.size());

    cv::Mat inverted_affine_matrix;
    cv::invertAffineTransform(affine_matrix, inverted_affine_matrix);

    const auto warpFunc = cvGS::warp<fk::WarpType::Affine, CV_8UC3>(d_img, inverted_affine_matrix, img.size());
    auto writeFunc = cvGS::write<CV_8UC3>(d_resultcvGS);
    cvGS::executeOperations(stream, warpFunc, fk::Cast<float3, uchar3>::build(), writeFunc);

    stream.waitForCompletion();

    // Download the result back to CPU
    cv::Mat resultcv(d_resultcv);
    cv::Mat resultcvGS(d_resultcvGS);

    return true;
}

int launch() {
    return testPerspective() && testAffine() ? 0 : -1;
}
