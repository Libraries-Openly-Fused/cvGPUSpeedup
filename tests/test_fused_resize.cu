#include "hip/hip_runtime.h"
/* Copyright 2023 Oscar Amoros Huguet

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <fstream>
#include <iostream>

#include "testsCommon.cuh"
#include <opencv2/opencv.hpp>
#include <cvGPUSpeedup.cuh>

int main() {
    constexpr size_t NUM_ELEMS_X = 6244;
    constexpr size_t NUM_ELEMS_Y = 4168;

    cv::cuda::Stream cv_stream;

    cv::Mat::setDefaultAllocator(cv::cuda::HostMem::getAllocator(cv::cuda::HostMem::AllocType::PAGE_LOCKED));

    const std::string filePath{ "C:/Users/oscar/Documents/GitHub/cvGPUSpeedup/images/raw6K.nv12" };
    // Open a binary file named "example.bin"
    std::ifstream file(filePath, std::ios::binary | std::ios::ate);
    std::streamsize size = file.tellg();
    file.seekg(0, std::ios::beg);
    char* buffer = new char[size];
    if (file.read(buffer, size)) {
        // use buffer
        hipStream_t stream;
        gpuErrchk(hipStreamCreate(&stream));

        constexpr fk::Size down(1920, 1080);
        cv::Mat h_result(down.height, down.width, CV_8UC4);
        cv::Mat nv12Image(cv::Size(NUM_ELEMS_X, NUM_ELEMS_Y + (NUM_ELEMS_Y/2)), CV_8UC1, buffer);

        uchar* d_dataSource;
        size_t sourcePitch;
        gpuErrchk(hipMallocPitch(&d_dataSource, &sourcePitch, NUM_ELEMS_X, NUM_ELEMS_Y + (NUM_ELEMS_Y / 2)));
        fk::RawPtr<fk::_2D, uchar> d_nv12Image{ d_dataSource, {(uint)NUM_ELEMS_X, (uint)NUM_ELEMS_Y, (uint)sourcePitch} };
        fk::Ptr2D<uchar4> d_rgbaImage(down.width, down.height);
        fk::Ptr2D<uchar4> d_rgbaImageBig(NUM_ELEMS_X, NUM_ELEMS_Y);

        gpuErrchk(hipMemcpy2DAsync(d_nv12Image.data, d_nv12Image.dims.pitch,
                          nv12Image.data, nv12Image.step,
                          NUM_ELEMS_X, NUM_ELEMS_Y + (NUM_ELEMS_Y / 2), hipMemcpyHostToDevice, stream));

        fk::Read<fk::ReadYUV<fk::NV12>> read { d_nv12Image, {NUM_ELEMS_X, NUM_ELEMS_Y} };
        fk::Unary<fk::ConvertYUVToRGB<fk::NV12, fk::Full, fk::bt709, true>> cvtColor {};
        fk::Write<fk::PerThreadWrite<fk::_2D, uchar4>> write { d_rgbaImageBig.ptr() };
        fk::executeOperations(stream, read, cvtColor, write);

        fk::Read<fk::PerThreadRead<fk::_2D, uchar4>> read2{ d_rgbaImageBig.ptr(), {NUM_ELEMS_X, NUM_ELEMS_Y} };
        fk::Unary<fk::VectorReorder<uchar4, 2, 1, 0, 3>> cvtColor2{};
        fk::Write<fk::PerThreadWrite<fk::_2D, uchar4>> write2{ d_rgbaImageBig.ptr() };
        fk::executeOperations(stream, read2, cvtColor2, write2);

        auto read3 = fk::resize<uchar4, fk::INTER_LINEAR>(d_rgbaImageBig.ptr(), down, 0., 0.);
        fk::Unary<fk::SaturateCast<float4, uchar4>> convertTo3 {};
        fk::Write<fk::PerThreadWrite<fk::_2D, uchar4>> write3 { d_rgbaImage.ptr() };
        fk::executeOperations(stream, read3, convertTo3, write3);
        gpuErrchk(hipMemcpy2DAsync(h_result.data, h_result.step,
                                    d_rgbaImage.ptr().data, d_rgbaImage.dims().pitch,
                                    down.width * sizeof(uchar4), down.height, hipMemcpyDeviceToHost, stream));
        gpuErrchk(hipStreamSynchronize(stream));

        using PixelReadOp = fk::ComposedOperation<fk::Read<fk::ReadYUV<fk::NV12>>, fk::Unary<fk::ConvertYUVToRGB<fk::NV12, fk::Full, fk::bt709, true>>>;
        fk::Binary<PixelReadOp> readOpInstance{ { {d_nv12Image, {}}, {} } };
        auto imgSize = d_nv12Image.dims;
        auto readOp = fk::resize<PixelReadOp, fk::INTER_LINEAR>(readOpInstance.params, fk::Size(imgSize.width, imgSize.height), down);
        auto convertOp = fk::Unary<fk::SaturateCast<float4, uchar4>>{};
        auto colorConvert = fk::Unary<fk::VectorReorder<uchar4, 2, 1, 0, 3>>{};
        auto writeOp = fk::Write<fk::PerThreadWrite<fk::_2D, uchar4>>{ d_rgbaImage.ptr() };
        fk::executeOperations(stream, readOp, convertOp, colorConvert, writeOp);
        gpuErrchk(hipMemcpy2DAsync(h_result.data, h_result.step,
                                    d_rgbaImage.ptr().data, d_rgbaImage.dims().pitch,
                                    down.width * sizeof(uchar4), down.height, hipMemcpyDeviceToHost, stream));

        gpuErrchk(hipStreamSynchronize(stream));

        gpuErrchk(hipFree(d_dataSource));

        gpuErrchk(hipStreamDestroy(stream));

    } else {
        // Print an error message if the file cannot be opened
        std::cerr << "Error: cannot open file\n";
    }
    file.close();
    delete buffer;

    return 0;
}