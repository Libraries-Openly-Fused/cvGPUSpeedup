#include "hip/hip_runtime.h"
/* Copyright 2025 Oscar Amoros Huguet

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "tests/testsCommon.cuh"
#include <cvGPUSpeedup.cuh>
#include <opencv2/cudaimgproc.hpp>

#include "tests/main.h"

#ifdef ENABLE_BENCHMARK
constexpr char VARIABLE_DIMENSION[]{ "Batch size" };

#ifndef CUDART_MAJOR_VERSION
#error CUDART_MAJOR_VERSION Undefined!
#elif (CUDART_MAJOR_VERSION == 11)
constexpr size_t NUM_EXPERIMENTS = 8;
#elif (CUDART_MAJOR_VERSION == 12)
constexpr size_t NUM_EXPERIMENTS = 16;
#endif // CUDART_MAJOR_VERSION

constexpr size_t FIRST_VALUE = 1;
constexpr size_t INCREMENT = 10;
constexpr std::array<size_t, NUM_EXPERIMENTS> batchValues = arrayIndexSecuence<FIRST_VALUE, INCREMENT, NUM_EXPERIMENTS>;

template <int CV_TYPE_I, int CV_TYPE_O, int BATCH>
bool test_cpu_batchresize_x_split3D(size_t NUM_ELEMS_X, size_t NUM_ELEMS_Y, cv::cuda::Stream& cv_stream, bool enabled) {
    std::stringstream error_s;
    bool passed = true;
    bool exception = false;

    if (enabled) {
        struct Parameters {
            cv::Scalar init;
            cv::Scalar alpha;
            cv::Scalar val_sub;
            cv::Scalar val_div;
        };

        double alpha = 0.3;

        std::vector<Parameters> params = {
            {{2u}, {alpha}, {1.f}, {3.2f}},
            {{2u, 37u}, {alpha, alpha}, {1.f, 4.f}, {3.2f, 0.6f}},
            {{5u, 5u, 5u}, {alpha, alpha, alpha}, {1.f, 4.f, 3.2f}, {3.2f, 0.6f, 11.8f}},
            {{2u, 37u, 128u, 20u}, {alpha, alpha, alpha, alpha}, {1.f, 4.f, 3.2f, 0.5f}, {3.2f, 0.6f, 11.8f, 33.f}} };

        cv::Scalar val_init = params.at(CV_MAT_CN(CV_TYPE_O) - 1).init;
        cv::Scalar val_alpha = params.at(CV_MAT_CN(CV_TYPE_O) - 1).alpha;
        cv::Scalar val_sub = params.at(CV_MAT_CN(CV_TYPE_O) - 1).val_sub;
        cv::Scalar val_div = params.at(CV_MAT_CN(CV_TYPE_O) - 1).val_div;

        constexpr int CROP_WIDTH = 60;
        constexpr int CROP_HEIGHT = 120;

        try {
            cv::cuda::GpuMat d_input((int)NUM_ELEMS_Y, (int)NUM_ELEMS_X, CV_TYPE_I, val_init);
            std::array<cv::Rect2d, BATCH> crops_2d;
            for (int crop_i = 0; crop_i < BATCH; crop_i++) {
                crops_2d[crop_i] =
                    cv::Rect2d(cv::Point2d(crop_i, crop_i), cv::Point2d(crop_i + CROP_WIDTH, crop_i + CROP_HEIGHT));
            }

            cv::Size up(64, 128);
            cv::cuda::GpuMat d_up(up, CV_TYPE_O);
            cv::cuda::GpuMat d_temp(up, CV_TYPE_O);
            cv::cuda::GpuMat d_temp2(up, CV_TYPE_O);

            std::array<std::vector<cv::cuda::GpuMat>, BATCH> d_output_cv;
            std::array<std::vector<cv::cuda::GpuMat>, BATCH> d_output_cvGS;
            std::array<std::vector<cv::Mat>, BATCH> h_cvResults;
            std::array<std::vector<cv::Mat>, BATCH> h_cvGSResults;

            cv::cuda::GpuMat d_tensor_output(BATCH, up.width * up.height * CV_MAT_CN(CV_TYPE_O), CV_MAT_DEPTH(CV_TYPE_O));
            d_tensor_output.step = up.width * up.height * CV_MAT_CN(CV_TYPE_O) * sizeof(BASE_CUDA_T(CV_TYPE_O));

            cv::Mat diff(up, CV_MAT_DEPTH(CV_TYPE_O));
            cv::Mat h_tensor_output(BATCH, up.width * up.height * CV_MAT_CN(CV_TYPE_O), CV_MAT_DEPTH(CV_TYPE_O));

            std::array<cv::cuda::GpuMat, BATCH> crops;
            cv::cuda::GpuMat crop_32F(cv::Size(CROP_WIDTH, CROP_HEIGHT), CV_32FC3);
            for (int crop_i = 0; crop_i < BATCH; crop_i++) {
                crops[crop_i] = d_input(crops_2d[crop_i]);
                for (int i = 0; i < CV_MAT_CN(CV_TYPE_I); i++) {
                    d_output_cv.at(crop_i).emplace_back(up, CV_MAT_DEPTH(CV_TYPE_O));
                    h_cvResults.at(crop_i).emplace_back(up, CV_MAT_DEPTH(CV_TYPE_O));
                }
            }

            constexpr bool correctDept = CV_MAT_DEPTH(CV_TYPE_O) == CV_32F;

            std::cout << "Executing " << __func__ << " fusing " << BATCH << " operations. " << ((BATCH - FIRST_VALUE) / INCREMENT)+1 << "/" << NUM_EXPERIMENTS << std::endl;
            BenchmarkResultsNumbers resF;
            resF.OCVelapsedTimeMax = fk::minValue<float>;
            resF.OCVelapsedTimeMin = fk::maxValue<float>;
            resF.OCVelapsedTimeAcum = 0.f;
            resF.cvGSelapsedTimeMax = fk::minValue<float>;
            resF.cvGSelapsedTimeMin = fk::maxValue<float>;
            resF.cvGSelapsedTimeAcum = 0.f;
            hipStream_t stream = cv::cuda::StreamAccessor::getStream(cv_stream);
            std::array<float, ITERS> OCVelapsedTime;
            std::array<float, ITERS> cvGSelapsedTime;
            for (int i = 0; i < ITERS; i++) {
                // OpenCV version
                const auto cpu_start1 = std::chrono::high_resolution_clock::now();
                for (int crop_i = 0; crop_i < BATCH; crop_i++) {
                    crops[crop_i].convertTo(crop_32F, CV_TYPE_O, 1, cv_stream);
                    cv::cuda::resize(crop_32F, d_up, up, 0., 0., cv::INTER_LINEAR, cv_stream);
                    cv::cuda::multiply(d_up, val_alpha, d_temp, 1.0, -1, cv_stream);
                    if constexpr (CV_MAT_CN(CV_TYPE_I) == 3 && correctDept) {
                        cv::cuda::cvtColor(d_temp, d_temp, cv::COLOR_RGB2BGR, 0, cv_stream);
                    } else if constexpr (CV_MAT_CN(CV_TYPE_I) == 4 && correctDept) {
                        cv::cuda::cvtColor(d_temp, d_temp, cv::COLOR_RGBA2BGRA, 0, cv_stream);
                    }
                    cv::cuda::subtract(d_temp, val_sub, d_temp2, cv::noArray(), -1, cv_stream);
                    cv::cuda::divide(d_temp2, val_div, d_temp, 1.0, -1, cv_stream);
                    cv::cuda::split(d_temp, d_output_cv[crop_i], cv_stream);
                }
                const auto cpu_end1 = std::chrono::high_resolution_clock::now();
                std::chrono::duration<float, std::milli> cpu_elapsed1 = cpu_end1 - cpu_start1;
                OCVelapsedTime[i] = cpu_elapsed1.count();
                resF.OCVelapsedTimeMax = resF.OCVelapsedTimeMax < OCVelapsedTime[i] ? OCVelapsedTime[i] : resF.OCVelapsedTimeMax;
                resF.OCVelapsedTimeMin = resF.OCVelapsedTimeMin > OCVelapsedTime[i] ? OCVelapsedTime[i] : resF.OCVelapsedTimeMin;
                resF.OCVelapsedTimeAcum += OCVelapsedTime[i];

                // cvGPUSpeedup
                const auto cpu_start = std::chrono::high_resolution_clock::now();
                cvGS::executeOperations(cv_stream, cvGS::resize<CV_TYPE_I, cv::INTER_LINEAR, BATCH>(crops, up, BATCH),
                    cvGS::cvtColor<cv::COLOR_RGB2BGR, CV_TYPE_O>(), cvGS::multiply<CV_TYPE_O>(val_alpha),
                    cvGS::subtract<CV_TYPE_O>(val_sub), cvGS::divide<CV_TYPE_O>(val_div),
                    cvGS::split<CV_TYPE_O>(d_tensor_output, up));
                const auto cpu_end = std::chrono::high_resolution_clock::now();
                std::chrono::duration<float, std::milli> cpu_elapsed = cpu_end - cpu_start;

                cvGSelapsedTime[i] = cpu_elapsed.count();
                resF.cvGSelapsedTimeMax = resF.cvGSelapsedTimeMax < cvGSelapsedTime[i] ? cvGSelapsedTime[i] : resF.cvGSelapsedTimeMax;
                resF.cvGSelapsedTimeMin = resF.cvGSelapsedTimeMin > cvGSelapsedTime[i] ? cvGSelapsedTime[i] : resF.cvGSelapsedTimeMin;
                resF.cvGSelapsedTimeAcum += cvGSelapsedTime[i]; 
                if (warmup) break;
            }
            processExecution<CV_TYPE_I, CV_TYPE_O, BATCH, ITERS, batchValues.size(), batchValues>(resF, __func__, OCVelapsedTime, cvGSelapsedTime, VARIABLE_DIMENSION);
                    d_tensor_output.download(h_tensor_output, cv_stream);

            // Verify results
            for (int crop_i = 0; crop_i < BATCH; crop_i++) {
                for (int i = 0; i < CV_MAT_CN(CV_TYPE_O); i++) {
                    d_output_cv[crop_i].at(i).download(h_cvResults[crop_i].at(i), cv_stream);
                }
            }

            cv_stream.waitForCompletion();

            for (int crop_i = 0; crop_i < BATCH; crop_i++) {
                cv::Mat row = h_tensor_output.row(crop_i);
                for (int i = 0; i < CV_MAT_CN(CV_TYPE_O); i++) {
                    int planeStart = i * up.width * up.height;
                    int planeEnd = ((i + 1) * up.width * up.height) - 1;
                    cv::Mat plane = row.colRange(planeStart, planeEnd);
                    h_cvGSResults[crop_i].push_back(cv::Mat(up.height, up.width, plane.type(), plane.data));
                }
            }

            for (int crop_i = 0; crop_i < BATCH; crop_i++) {
                for (int i = 0; i < CV_MAT_CN(CV_TYPE_O); i++) {
                    cv::Mat cvRes = h_cvResults[crop_i].at(i);
                    cv::Mat cvGSRes = h_cvGSResults[crop_i].at(i);
                    diff = cv::abs(cvRes - cvGSRes);
                    bool passedThisTime = checkResults<CV_MAT_DEPTH(CV_TYPE_O)>(diff.cols, diff.rows, diff);
                    passed &= passedThisTime;
                }
            }
        }
        catch (const cv::Exception& e) {
            if (e.code != -210) {
                error_s << e.what();
                passed = false;
                exception = true;
            }
        }
        catch (const std::exception& e) {
            error_s << e.what();
            passed = false;
            exception = true;
        }

        if (!passed) {
            if (!exception) {
                std::stringstream ss;
                ss << "test_batchresize_x_split3D<" << cvTypeToString<CV_TYPE_I>() << ", " << cvTypeToString<CV_TYPE_O>();
                std::cout << ss.str() << "> failed!! RESULT ERROR: Some results do not match baseline." << std::endl;
            } else {
                std::stringstream ss;
                ss << "test_batchresize_x_split3D<" << cvTypeToString<CV_TYPE_I>() << ", " << cvTypeToString<CV_TYPE_O>();
                std::cout << ss.str() << "> failed!! EXCEPTION: " << error_s.str() << std::endl;
            }
        }
    }

    return passed;
}

template <int CV_TYPE_I, int CV_TYPE_O, size_t... Is>
bool test_cpu_batchresize_x_split3D(const size_t NUM_ELEMS_X, const size_t NUM_ELEMS_Y, std::index_sequence<Is...> seq,
    cv::cuda::Stream cv_stream, bool enabled) {
    bool passed = true;
    int dummy[] = { (passed &= test_cpu_batchresize_x_split3D<CV_TYPE_I, CV_TYPE_O, batchValues[Is]>(NUM_ELEMS_X, NUM_ELEMS_Y,
                                                                                                cv_stream, enabled),
                    0)... };
    return passed;
}

#endif // ENABLE_BENCHMARK

int launch() {
#ifdef ENABLE_BENCHMARK
    constexpr size_t NUM_ELEMS_X = 3840;
    constexpr size_t NUM_ELEMS_Y = 2160;

    cv::cuda::Stream cv_stream;

    cv::Mat::setDefaultAllocator(cv::cuda::HostMem::getAllocator(cv::cuda::HostMem::AllocType::PAGE_LOCKED));

    std::unordered_map<std::string, bool> results;
    results["test_cpu_batchresize_x_split3D"] = true;
    std::make_index_sequence<batchValues.size()> iSeq{};

#define LAUNCH_TESTS(CV_INPUT, CV_OUTPUT)                                                                              \
  results["test_cpu_batchresize_x_split3D"] &=                                                                             \
      test_cpu_batchresize_x_split3D<CV_INPUT, CV_OUTPUT>(NUM_ELEMS_X, NUM_ELEMS_Y, iSeq, cv_stream, true);

    // Warming up for the benchmarks
    warmup = true;
    LAUNCH_TESTS(CV_8UC3, CV_32FC3)
    warmup = false;

    LAUNCH_TESTS(CV_8UC3, CV_32FC3)

#undef LAUNCH_TESTS

    for (auto&& [_, file] : currentFile) {
        file.close();
    }

    int returnValue = 0;
    for (const auto& [key, passed] : results) {
        if (passed) {
            std::cout << key << " passed!!" << std::endl;
        } else {
            std::cout << key << " failed!!" << std::endl;
            returnValue = -1;
        }
    }

    return returnValue;
#else
    return 0;
#endif // ENABLE_BENCHMARK
}