/* Copyright 2025 Oscar Amoros Huguet

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <benchmarks/opencv/verticalfusion/vertical_fusion_kernel_instances/mul1/mul3102-4002.h>
#include <benchmarks/opencv/verticalfusion/vertical_fusion_kernel_instances/mul1/launchMulMacro.cuh>

LAUNCH(3102)
LAUNCH(3202)
LAUNCH(3302)
LAUNCH(3402)
LAUNCH(3502)
LAUNCH(3602)
LAUNCH(3702)
LAUNCH(3802)
LAUNCH(3902)
LAUNCH(4002)
