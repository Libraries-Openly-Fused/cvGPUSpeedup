/* Copyright 2025 Oscar Amoros Huguet

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <benchmarks/opencv/verticalfusion/vertical_fusion_kernel_instances/mul1/mul10102-11002.h>
#include <benchmarks/opencv/verticalfusion/vertical_fusion_kernel_instances/mul1/launchMulMacro.cuh>

LAUNCH(10102)
LAUNCH(10202)
LAUNCH(10302)
LAUNCH(10402)
LAUNCH(10502)
LAUNCH(10602)
LAUNCH(10702)
LAUNCH(10802)
LAUNCH(10902)
LAUNCH(11002)
