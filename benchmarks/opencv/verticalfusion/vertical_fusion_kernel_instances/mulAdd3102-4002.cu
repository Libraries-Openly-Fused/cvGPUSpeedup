/* Copyright 2025 Oscar Amoros Huguet

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <benchmarks/opencv/verticalfusion/vertical_fusion_kernel_instances/mul2-1002.h>
#include <benchmarks/opencv/verticalfusion/vertical_fusion_static_loop.cuh>

template <int CV_TYPE_I, int CV_TYPE_O, int OPS_PER_ITER, size_t NumOps, typename DeviceFunction>
void launchVerticalFusion(const std::array<cv::cuda::GpuMat, 50>& crops,
    const cv::cuda::Stream& cv_stream,
    const float& alpha,
    const cv::cuda::GpuMat& d_tensor_output,
    const cv::Size& cropSize,
    const DeviceFunction& dFunc) {
    VerticalFusion<CV_8UC1, CV_32FC1, 2, NumOps, DeviceFunction>::execute(crops, cv_stream, alpha, d_tensor_output, cropSize, dFunc);
}

#define LAUNCH(NumOps) \
void launchMulAdd##NumOps(const std::array<cv::cuda::GpuMat, 50>& crops, \
    const cv::cuda::Stream& cv_stream, \
    const float& alpha, \
    const cv::cuda::GpuMat& d_tensor_output, \
    const cv::Size& cropSize, \
    const MulFuncType& dFunc) { \
    launchVerticalFusion<CV_8UC1, CV_32FC1, 2, NumOps, MulFuncType>(crops, cv_stream, alpha, d_tensor_output, cropSize, dFunc); \
}

LAUNCH(3102)
LAUNCH(3202)
LAUNCH(3302)
LAUNCH(3402)
LAUNCH(3502)
LAUNCH(3602)
LAUNCH(3702)
LAUNCH(3802)
LAUNCH(3902)
LAUNCH(4002)

#undef LAUNCH